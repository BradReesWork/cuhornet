#include "Hornet.hpp"
#include "StandardAPI.hpp"
#include "Util/BatchFunctions.hpp"
#include <Host/FileUtil.hpp>            //xlib::extract_filepath_noextension
#include <Device/Util/CudaUtil.cuh>     //xlib::deviceInfo
#include <algorithm>                    //std:.generate
#include <chrono>                       //std::chrono
#include <random>                       //std::mt19937_64
#include <hip/hip_runtime_api.h>

//using namespace hornets_nest;
using namespace timer;
using namespace std::string_literals;

using vert_t = int;
using Init = hornet::HornetInit<vert_t, hornet::EMPTY, hornet::TypeList<int, float>>;
using HornetGPU = hornet::gpu::Hornet<vert_t, hornet::EMPTY, hornet::TypeList<int, float>>;
using UpdatePtr = hornet::BatchUpdatePtr<vert_t, hornet::TypeList<int, float>, hornet::DeviceType::HOST>;
using Update = hornet::gpu::BatchUpdate<vert_t, hornet::TypeList<int, float>>;

/**
 * @brief Example tester for Hornet
 */
int exec(int argc, char* argv[]) {
    using namespace graph::structure_prop;
    using namespace graph::parsing_prop;
    xlib::device_info();

    graph::GraphStd<vert_t, vert_t> graph;
    graph.read(argv[1]);
    //--------------------------------------------------------------------------
    Init hornet_init(graph.nV(), graph.nE(), graph.csr_out_offsets(),
                           graph.csr_out_edges());

    //Use meta with hornet_init
    std::vector<int> edge_meta_0(graph.nE(), 0);
    std::vector<float> edge_meta_1(graph.nE(), 1);
    hornet_init.insertEdgeData(edge_meta_0.data(), edge_meta_1.data());

    HornetGPU hornet_gpu(hornet_init);
    using namespace hornets_nest::batch_gen_property;
    using namespace hornets_nest::host;

    vert_t* batch_src, *batch_dst;
    int batch_size = std::stoi(argv[2]);

    allocatePageLocked(batch_src, batch_size);
    allocatePageLocked(batch_dst, batch_size);
    //std::vector<int> batch_edge_meta_0(batch_size, 2);
    std::vector<float> batch_edge_meta_1(batch_size, -1.5);

    generateBatch(graph,
            batch_size, batch_src, batch_dst,
            hornets_nest::BatchGenType::INSERT);
    //UpdatePtr ptr(batch_size, batch_src, batch_dst, batch_edge_meta_0.data(), batch_edge_meta_1.data());
    UpdatePtr ptr(batch_size, batch_src, batch_dst, nullptr, batch_edge_meta_1.data());

    Update batch_update(ptr);

    printf("ne: %d\n", hornet_gpu.nE());
    std::cout<<"=======\n";
    Timer<DEVICE> TM(3);
    TM.start();
    hornet_gpu.insert(batch_update);

    TM.stop();

    printf("ne: %d\n", hornet_gpu.nE());
    std::cout<<"=======\n";
    TM.print("Insertion " + std::to_string(batch_size) + ":  ");

    freePageLocked(batch_dst, batch_src);

    return 0;
}

int main(int argc, char* argv[]) {
    int ret = 0;
#if defined(RMM_WRAPPER)
    gpu::initializeRMMPoolAllocation();//update initPoolSize if you know your memory requirement and memory availability in your system, if initial pool size is set to 0 (default value), RMM currently assigns half the device memory.
    {//scoping technique to make sure that gpu::finalizeRMMPoolAllocation is called after freeing all RMM allocations.
#endif

    ret = exec(argc, argv);

#if defined(RMM_WRAPPER)
    }//scoping technique to make sure that gpu::finalizeRMMPoolAllocation is called after freeing all RMM allocations.
    gpu::finalizeRMMPoolAllocation();
#endif

    return ret;
}

