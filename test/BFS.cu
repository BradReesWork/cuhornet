#include "hip/hip_runtime.h"
#include "cuStingerAlg/cuStingerAlg.cuh"    //cuStingerAlg
#include "cuStingerAlg/LoadBalancing/BinarySearch.cuh"
#include "cuStingerAlg/LoadBalancing/VertexBased.cuh"
#include "cuStingerAlg/Operator.cuh"        //Operator
#include "cuStingerAlg/Queue/TwoLevelQueue.cuh"   //Queue

#include <GraphIO/BFS.hpp>              //BFS
#include <GraphIO/GraphStd.hpp>         //GraphStd
#include <Support/Device/Algorithm.cuh> //cu::equal
#include <Support/Host/Timer.hpp>       //Timer

using namespace custinger_alg;
using namespace timer;
using namespace load_balacing;
using namespace custinger;

using dist_t = int;
const dist_t INF = std::numeric_limits<dist_t>::max();

struct BFSData {
    BFSData(size_t allocation) : queue(allocation)  {}

    TwoLevelQueue<vid_t> queue;
    dist_t*              d_distances;
    dist_t               level = 1;
};

__device__ __forceinline__
void VertexInit(vid_t index, void* optional_field) {
    auto bfs_data = *reinterpret_cast<BFSData*>(optional_field);
    bfs_data.d_distances[index] = INF;
}

__device__ __forceinline__
void BFSOperatorAtomic(Vertex src, Edge edge, void* optional_field) {
    auto bfs_data = *reinterpret_cast<BFSData*>(optional_field);
    auto dst = edge.dst();
    auto old = atomicCAS(bfs_data.d_distances + dst, INF, bfs_data.level);
    if (old == INF)
        bfs_data.queue.insert(src.id());     // the vertex dst is active*/
}

__device__ __forceinline__
void BFSOperatorNoAtomic(Vertex src, Edge edge, void* optional_field) {
    auto bfs_data = *reinterpret_cast<BFSData*>(optional_field);
    auto dst = edge.dst();
    if (bfs_data.d_distances[dst] == INF) {
        bfs_data.d_distances[dst] = bfs_data.level;
        bfs_data.queue.insert(src.id());    // the vertex dst is active
    }
}

//==============================================================================

int main(int argc, char* argv[]) {
    using namespace custinger;
    hipSetDevice(1);
    vid_t bfs_source = 0;
    //--------------------------------------------------------------------------
    //////////////
    // HOST BFS //
    //////////////
    graph::GraphStd<vid_t, eoff_t> graph;
    graph.read(argv[1]);
    graph::BFS<vid_t, eoff_t> bfs(graph);
    bfs.run(bfs_source);

    auto h_distances = bfs.distances();
    //--------------------------------------------------------------------------
    /////////////////
    // DEVICE INIT //
    /////////////////
    cuStingerInit custinger_init(graph.nV(), graph.nE(), graph.out_offsets(),
                                 graph.out_edges());

    cuStinger custiger_graph(custinger_init);

    dist_t* d_distances;
    Allocate alloc(d_distances, graph.nV());
    //--------------------------------------------------------------------------
    //////////////
    // BFS INIT //
    //////////////
    forAllnumV<VertexInit>(custiger_graph, d_distances);
    cuMemcpyToDevice(0, d_distances + bfs_source);
    //TwoLevelQueue<vid_t> queue(graph.nV() * 2);
    //queue.insert(bfs_source);

    load_balacing::BinarySearch lb(graph.out_offsets(), graph.nV());
    //load_balacing::VertexBased lb;

    BFSData bfs_data(graph.nV() * 2);
    bfs_data.queue.insert(bfs_source);

    Timer<DEVICE> TM;
    TM.start();
    //--------------------------------------------------------------------------
    ///////////////////
    // BFS ALGORITHM //
    ///////////////////
    while (bfs_data.queue.size() > 0) {
        lb.traverse_edges<BFSOperatorNoAtomic>((void*) bfs_data);
        bfs_data.queue.swap();
        bfs_data.level++;
    }
    //--------------------------------------------------------------------------
    ////////////////////
    // BFS VALIDATION //
    ////////////////////
    TM.stop();
    TM.print("BFS");

    auto is_correct = cu::equal(h_distances, h_distances + graph.nV(),
                                d_distances);
    std::cout << (is_correct ? "\nCorrect <>\n\n" : "\n! Not Correct\n\n");
}
