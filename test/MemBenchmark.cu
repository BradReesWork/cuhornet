#include "hip/hip_runtime.h"
#include <Device/SafeCudaAPI.cuh>
#include <Device/SimpleKernels.cuh>
#include <Device/Timer.cuh>
#include <cmath>
#include <limits>
#include <iomanip>
#include <iostream>
#include <vector>

using namespace timer;
using xlib::byte_t;
using ttime_t = float;

int main() {
    size_t size = 1024;
    Timer<DEVICE> TM;

    std::vector<ttime_t> allocation_time;
    std::vector<ttime_t> allocation_pinned_time;
    std::vector<ttime_t> H2D_time;
    std::vector<ttime_t> H2D_pinned_time;
    std::vector<ttime_t> D2D_time;
    std::vector<ttime_t> memset_time;
    byte_t* d_array, *h_array_pinned;

    std::cout << "Computing";

    while (true) {
        std::cout << "." << std::flush;
        //======================================================================
        TM.start();

        if (hipMalloc(&d_array, size) != hipSuccess)
            break;

        TM.stop();
        allocation_time.push_back(TM.duration());
        //----------------------------------------------------------------------
        auto h_array = new byte_t[size];
        TM.start();

        cuMemcpyToDevice(h_array, size, d_array);

        TM.stop();
        delete[] h_array;
        H2D_time.push_back(TM.duration());
        //----------------------------------------------------------------------
        TM.start();

        hipHostMalloc(&h_array_pinned, size);

        TM.stop();

        allocation_pinned_time.push_back(TM.duration());
        //----------------------------------------------------------------------
        TM.start();

        cuMemcpyToDeviceAsync(h_array_pinned, size, d_array);

        TM.stop();
        hipHostFree(h_array_pinned);
        H2D_pinned_time.push_back(TM.duration());
        //----------------------------------------------------------------------
        TM.start();

        hipMemset(d_array, 0x00, size);

        TM.stop();
        memset_time.push_back(TM.duration());
        //----------------------------------------------------------------------
        byte_t* d_array2;
        if (hipMalloc(&d_array2, size) == hipSuccess) {
            TM.start();

            hipMemcpy(d_array2, d_array, size, hipMemcpyDeviceToDevice);

            TM.stop();
            D2D_time.push_back(TM.duration());
            cuFree(d_array2);
        }
        else {
            D2D_time.push_back(std::nan(""));
        }
        cuFree(d_array);
        //----------------------------------------------------------------------
        size *= 2;
    }
    size = 1024;
    std::cout << "\n\n" << std::setprecision(2) << std::right << std::fixed
              << std::setw(8)  << "SIZE"
              << std::setw(12) << "hipMalloc"
              << std::setw(18) << "cudaMallocPinned"
              << std::setw(11) << "MemcpyHtD"
              << std::setw(14) << "MemcpyHtDPin"
              << std::setw(11) << "MemcpyDtD"
              << std::setw(8)  << "Memset" << std::endl;
    xlib::char_sequence('-', 80);

    for (size_t i = 0; i < H2D_time.size(); i++) {
        std::cout << std::setw(8)  << xlib::human_readable(size)
                  << std::setw(12) << allocation_time[i]
                  << std::setw(18) << allocation_pinned_time[i]
                  << std::setw(11) << H2D_time[i]
                  << std::setw(14) << H2D_pinned_time[i]
                  << std::setw(11) << D2D_time[i]
                  << std::setw(8)  << memset_time[i] << "\n";
        size *= 2;
    }
    //==========================================================================
    Timer<DEVICE> TM2(2);

    xlib::byte_t array[4 * xlib::MB];
    hipMalloc(&d_array, 4 * xlib::MB);
    hipHostMalloc(&h_array_pinned, 4 * xlib::MB);

    TM2.start();

    hipMemcpy(array, d_array, 4 * xlib::MB, hipMemcpyDeviceToHost);

    TM2.stop();
    TM2.print("Stack");

    TM2.start();

    hipMemcpyAsync(h_array_pinned, d_array, 4 * xlib::MB,
                    hipMemcpyDeviceToHost);

    TM2.stop();
    TM2.print("Pinned");

    hipFree(d_array);
    hipHostFree(h_array_pinned);
}
