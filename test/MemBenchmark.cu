#include "hip/hip_runtime.h"
#include <Device/SafeCudaAPI.cuh>
#include <Device/SimpleKernels.cuh>
#include <Device/Timer.cuh>
#include <cmath>
#include <limits>
#include <iomanip>
#include <iostream>
#include <vector>

using namespace timer;
using xlib::byte_t;

int main() {
    auto seed = std::chrono::high_resolution_clock::now().time_since_epoch()
                .count();
    std::mt19937_64 gen(seed);
    //std::generate(v.begin(), v.end(), std::rand);
    std::uniform_int_distribution<unsigned char>
        distribution(0, std::numeric_limits<unsigned char>::max());

    size_t size = 1024;
    Timer<DEVICE> TM;

    std::vector<float> H2D_time;
    std::vector<float> H2D_pinned_time;
    std::vector<float> D2D_time;
    std::vector<float> memcpy_kernel_time;
    std::vector<float> memset_time;
    std::vector<float> memset_kernel_time;
    std::cout << "Computing";

    while (true) {
        std::cout << "." << std::flush;
        //======================================================================
        byte_t* d_array;
        if (hipMalloc(&d_array, size) != hipSuccess)
            break;

        auto h_array = new byte_t[size];
        TM.start();

        cuMemcpyToDevice(h_array, size, d_array);

        TM.stop();
        delete[] h_array;
        H2D_time.push_back(TM.duration());
        //----------------------------------------------------------------------
        byte_t* h_array_pinned;
        hipHostMalloc(&h_array_pinned, size);
        TM.start();

        cuMemcpyToDevice(h_array_pinned, size, d_array);

        TM.stop();
        hipHostFree(h_array_pinned);
        H2D_pinned_time.push_back(TM.duration());
        //----------------------------------------------------------------------
        TM.start();

        hipMemset(d_array, 0x00, size);

        TM.stop();
        memset_time.push_back(TM.duration());
        //----------------------------------------------------------------------
        TM.start();

        cu::memset(reinterpret_cast<unsigned char*>(d_array), size,
                   (unsigned char) 0);

        TM.stop();
        CHECK_CUDA_ERROR
        memset_kernel_time.push_back(TM.duration());
        //----------------------------------------------------------------------
        byte_t* d_array2;
        if (hipMalloc(&d_array2, size) == hipSuccess) {
            TM.start();

            hipMemcpy(d_array2, d_array, size, hipMemcpyDeviceToDevice);

            TM.stop();
            D2D_time.push_back(TM.duration());
        //----------------------------------------------------------------------
            TM.start();

            cu::memcpy(d_array, size, d_array2);

            TM.stop();
            memcpy_kernel_time.push_back(TM.duration());
            cuFree(d_array2);
        }
        else {
            D2D_time.push_back(std::nan(""));
            memcpy_kernel_time.push_back(std::nan(""));
        }
        cuFree(d_array);
        //----------------------------------------------------------------------
        size *= 2;
    }
    size = 1024;
    std::cout << "\n\n" << std::setprecision(2) << std::right << std::fixed
              << std::setw(8)  << "SIZE"
              << std::setw(11) << "MemcpyHtD"
              << std::setw(14) << "MemcpyHtDPin"
              << std::setw(11) << "MemcpyDtD"
              << std::setw(14) << "MemcpyKernel"
              << std::setw(8)  << "Memset"
              << std::setw(14) << "MemsetKernel" << std::endl;
    xlib::char_sequence('-', 80);

    for (size_t i = 0; i < H2D_time.size(); i++) {
        std::cout << std::setw(8)  << xlib::human_readable(size)
                  << std::setw(11) << H2D_time[i]
                  << std::setw(14) << H2D_pinned_time[i]
                  << std::setw(11) << D2D_time[i]
                  << std::setw(14) << memcpy_kernel_time[i]
                  << std::setw(8)  << memset_time[i]
                  << std::setw(14) << memset_kernel_time[i] << "\n";
        size *= 2;
    }
}
