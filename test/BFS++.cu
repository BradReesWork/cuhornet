#include "hip/hip_runtime.h"



#include "cuStingerAlg/DeviceQueue.cuh"
#include "cuStingerAlg/Operator.cuh"    //Operator



#include "cuStingerAlg/cuStingerAlg.cuh"      //cuStingerAlg
#include "cuStingerAlg/TwoLevelQueue.cuh"       //Queue

#include "GraphIO/BFS.hpp"              //BFS
#include "GraphIO/GraphStd.hpp"         //GraphStd
#include "Support/Device/Algorithm.cuh" //cu::equal
#include "Support/Host/Timer.hpp"       //Timer
#include "cuStingerAlg/LoadBalancing/BinarySearch.cuh"

//#include "Core/cuStinger.hpp"           //cuStingerInit, cuStinger
#include "Csr/Csr.hpp"           //cuStingerInit, cuStinger
#include "Csr/CsrTypes.cuh"           //cuStingerInit, cuStinger

//using namespace csr;
using namespace cu_stinger_alg;
using namespace timer;
using namespace load_balacing;

using dist_t = int;

struct VertexInit;
struct BFSOperatorAtomic;
struct BFSOperatorNoAtomic;

//==============================================================================

int main(int argc, char* argv[]) {
    using namespace cu_stinger;
    hipSetDevice(1);
    vid_t bfs_source = 0;

    //--------------------------------------------------------------------------
    //////////////
    // HOST BFS //
    //////////////
    graph::GraphStd<vid_t, eoff_t> graph;
    graph.read(argv[1]);
    graph::BFS<vid_t, eoff_t> bfs(graph);
    bfs.run(bfs_source);

    auto h_distances = bfs.distances();
    //--------------------------------------------------------------------------
    /////////////////
    // DEVICE INIT //
    /////////////////
    cuStingerInit custinger_init(graph.nV(), graph.nE(),
                                 graph.out_offsets_array(),
                                 graph.out_edges_array());

    cuStinger custiger_graph(custinger_init);
    //Csr csr_graph(custinger_init);

    dist_t* d_distances;
    Allocate alloc(d_distances, graph.nV());
    //--------------------------------------------------------------------------
    //////////////
    // BFS INIT //
    //////////////
    forAllnumV<VertexInit>(d_distances);
    cuMemcpyToDevice(0, d_distances + bfs_source);

    dist_t level = 1;
    TwoLevelQueue<vid_t> queue(graph.nV() * 2);
    queue.insert(bfs_source);
    //queue.insert(bfs_sources, num_sources);               // Multi-sources BFS
    load_balacing::BinarySearch lb(queue, graph.out_offsets_array());

    Timer<DEVICE> TM;
    TM.start();
    //--------------------------------------------------------------------------
    ///////////////////
    // BFS ALGORITHM //
    ///////////////////
    while (queue.size() > 0) {
        lb.traverse_edges<BFSOperatorNoAtomic>(d_distances, level);
        level++;
    }
    //--------------------------------------------------------------------------
    ////////////////////
    // BFS VALIDATION //
    ////////////////////
    TM.stop();
    TM.print("BFS");

    auto is_correct = cu::equal(h_distances, h_distances + graph.nV(),
                                d_distances);
    std::cout << (is_correct ? "\nCorrect <>\n\n" : "\n! Not Correct\n\n");
}

//------------------------------------------------------------------------------

using cu_stinger::Vertex;
using cu_stinger::Edge;

const dist_t INF = std::numeric_limits<dist_t>::max();

struct VertexInit {
    __device__ __forceinline__
    static void apply(dist_t& vertex_distance) {
        vertex_distance = INF;
    }
};

struct BFSOperatorAtomic {
    __device__ __forceinline__
    static void apply(Vertex src, Edge edge,
                      DeviceQueue<cu_stinger::vid_t>& queue,
                      dist_t* d_distances, dist_t level) {

        auto dst = edge.dst();
        auto old = atomicCAS(d_distances + dst, INF, level);
        if (old == INF)
            queue.insert(src.id());     // the vertex dst is active
    }
};

struct BFSOperatorNoAtomic {
    __device__ __forceinline__
    static void apply(Vertex src, Edge edge,
                      DeviceQueue<cu_stinger::vid_t>& queue,
                      dist_t* d_distances, dist_t level) {

        auto dst = edge.dst();
        if (d_distances[dst] == INF) {
            d_distances[dst] = level;
            queue.insert(src.id());    // the vertex dst is active
        }
    }
};

//------------------------------------------------------------------------------
//#include <hip/hip_runtime_api.h>
//hipProfilerStart();
//hipProfilerStop();
//xlib::printArray(h_distances, graph.nV(), "Host\n");
//xlib::printArray(tmp_distance, graph.nV(), "Device\n");

/*auto statistics = bfs.statistics(0);
int l = 1;
for (const auto& it : statistics)
    std::cout << l++ << "\t" << it[2] << std::endl;*/
