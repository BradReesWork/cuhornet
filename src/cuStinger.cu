#include "hip/hip_runtime.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "main.h"
#include "cuStinger.hpp"

using namespace std;

__global__ void devMakeGPUStinger(int32_t* d_off, int32_t* d_adj,
	int verticesPerThreadBlock,cuStinger* custing)
{
	int32_t** d_cuadj = custing->d_adj;
	int32_t* d_utilized = custing->d_utilized;

	int32_t v_init=blockIdx.x*verticesPerThreadBlock;
	for (int v_hat=0; v_hat<verticesPerThreadBlock; v_hat++){
		int32_t v=v_init+v_hat;
		if(v>=custing->nv)
			break;
		for(int32_t e=threadIdx.x; e<d_utilized[v]; e+=blockDim.x){
			d_cuadj[v][e]=d_adj[d_off[v]+e];
		}
	}
}


void cuStinger::initcuStinger(int32_t* h_off, int32_t* h_adj){
	int32_t* d_off = (int32_t*)allocDeviceArray(nv+1,sizeof(int32_t));
	int32_t* d_adj = (int32_t*)allocDeviceArray(ne,sizeof(int32_t));
	copyArrayHostToDevice(h_off,d_off,nv,sizeof(int32_t));
	copyArrayHostToDevice(h_adj,d_adj,ne,sizeof(int32_t));

	dim3 numBlocks(1, 1);
	int32_t threads=64;
	dim3 threadsPerBlock(threads, 1);

	numBlocks.x = ceil((float)nv/(float)threads);
	if (numBlocks.x>16000){
		numBlocks.x=16000;
	}	

	int32_t verticesPerThreadBlock = ceil(float(nv)/float(numBlocks.x-1));

	devMakeGPUStinger<<<numBlocks,threadsPerBlock>>>(d_off,d_adj,verticesPerThreadBlock, d_cuStinger);

	freeDeviceArray(d_adj);	
	freeDeviceArray(d_off);
}


template <int32_t blockSize>
__global__ void reduce(int32_t *g_idata, int32_t *g_odata, int32_t n)
{
	extern __shared__ int32_t sdata[];
	int32_t tid = threadIdx.x;
	int32_t i = blockIdx.x*(blockSize*2) + tid;
	int32_t gridSize = blockSize*2*gridDim.x;
	sdata[tid] = 0;
	while (i < n) { sdata[tid] += g_idata[i] + g_idata[i+blockSize]; i += gridSize; }
	__syncthreads();
	if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
	if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
	if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
	if (tid < 32) {
		if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
		if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
		if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
		if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
		if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
		if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
	}
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}


#define BLOCK_SIZE 512
__global__ void total(int32_t * input, int32_t * output, int32_t len) {
    //@@ Load a segment of the input vector into shared memory
    __shared__ int32_t partialSum[2 * BLOCK_SIZE];
    int32_t t = threadIdx.x, start = 2 * blockIdx.x * BLOCK_SIZE;
    if (start + t < len)
       partialSum[t] = input[start + t];
    else
       partialSum[t] = 0;
    if (start + BLOCK_SIZE + t < len)
       partialSum[BLOCK_SIZE + t] = input[start + BLOCK_SIZE + t];
    else
       partialSum[BLOCK_SIZE + t] = 0;
    //@@ Traverse the reduction tree
    for (int stride = BLOCK_SIZE; stride >= 1; stride >>= 1) {
       __syncthreads();
       if (t < stride)
          partialSum[t] += partialSum[t+stride];
    }
    //@@ Write the computed sum of the block to the output vector at the 
    //@@ correct index
    if (t == 0)
       output[blockIdx.x] = partialSum[0];
}


int32_t cuStinger::sumDeviceArray(int32_t* arr){
	int32_t numOutputElements = nv / (BLOCK_SIZE<<1);
    if (nv % (BLOCK_SIZE<<1)) {
        numOutputElements++;
    }

	int32_t* d_out = (int32_t*)allocDeviceArray(nv, sizeof(int32_t*));

	total<<<numOutputElements,512>>>(d_utilized,d_out,nv);

	int32_t* h_out = (int32_t*)allocHostArray(nv, sizeof(int32_t*));
	
	int32_t sum=0;
	copyArrayDeviceToHost(d_out, h_out, nv, sizeof(int32_t));
	for(int i=0; i<numOutputElements; i++){
		 // cout << h_out[i] << ", ";
		sum+=h_out[i];
	}
	freeHostArray(h_out);
	freeDeviceArray(d_out);	
	return sum;
}



__global__ void deviceCopyMultipleAdjacencies(cuStinger* custing, int32_t** d_newadj, 
	int32_t* requireUpdates, int32_t requireCount ,int32_t verticesPerThreadBlock)
{
	int32_t** d_cuadj = custing->d_adj;
	int32_t* d_utilized = custing->d_utilized;

	int32_t v_init=blockIdx.x*verticesPerThreadBlock;
	for (int v_hat=0; v_hat<verticesPerThreadBlock; v_hat++){
		int32_t v= requireUpdates[v_init+v_hat];
		if(v>=requireCount)
			break;
		for(int32_t e=threadIdx.x; e<d_utilized[v]; e+=blockDim.x){
			d_newadj[v][e] = d_cuadj[v][e];
			// d_cuadj[v][e] = d_cuadj[v][e];
		}
	}
}

void cuStinger::copyMultipleAdjacencies(int32_t** d_newadj, 
	int32_t* requireUpdates, int32_t requireCount){

	dim3 numBlocks(1, 1);
	int32_t threads=32;
	dim3 threadsPerBlock(threads, 1);

	numBlocks.x = ceil((float)requireCount);
	if (numBlocks.x>16000){
		numBlocks.x=16000;
	}	
	int32_t verticesPerThreadBlock = ceil(float(requireCount)/float(numBlocks.x-1));

	cout << "### " << requireCount << " , " <<  numBlocks.x << " , " << verticesPerThreadBlock << " ###"  << endl; 

	deviceCopyMultipleAdjacencies<<<numBlocks,threadsPerBlock>>>(d_cuStinger,
		d_newadj, requireUpdates, requireCount, verticesPerThreadBlock);
	checkLastCudaError("Error in the first update sweep");
}



