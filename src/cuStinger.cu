#include "hip/hip_runtime.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "main.hpp"

using namespace std;


// void initializeCuStinger(cuStingerConfig);


__global__ void devMakeGPUStinger(int32_t* d_off, int32_t* d_adj,
	int verticesPerThreadBlock,cuStinger* custing)
{
	int32_t** d_cuadj = custing->d_adj;
	length_t* d_utilized = custing->d_utilized;

	int32_t v_init=blockIdx.x*verticesPerThreadBlock;
	for (int v_hat=0; v_hat<verticesPerThreadBlock; v_hat++){
		int32_t v=v_init+v_hat;
		if(v>=custing->nv)
			break;
		for(int32_t e=threadIdx.x; e<d_utilized[v]; e+=blockDim.x){
			d_cuadj[v][e]=d_adj[d_off[v]+e];
		}
	}
}


void cuStinger::internalCSRcuStinger(length_t* h_off, vertexId_t* h_adj, int ne){
	length_t* d_off = (length_t*)allocDeviceArray(nv+1,sizeof(int32_t));
	vertexId_t* d_adj = (length_t*)allocDeviceArray(ne,sizeof(int32_t));
	copyArrayHostToDevice(h_off,d_off,nv,sizeof(length_t));
	copyArrayHostToDevice(h_adj,d_adj,ne,sizeof(vertexId_t));

	dim3 numBlocks(1, 1);
	int32_t threads=64;
	dim3 threadsPerBlock(threads, 1);

	numBlocks.x = ceil((float)nv/(float)threads);
	if (numBlocks.x>16000){
		numBlocks.x=16000;
	}	

	int32_t verticesPerThreadBlock = ceil(float(nv)/float(numBlocks.x-1));

	devMakeGPUStinger<<<numBlocks,threadsPerBlock>>>(d_off,d_adj,verticesPerThreadBlock, d_cuStinger);

	freeDeviceArray(d_adj);	
	freeDeviceArray(d_off);
}



#define SUM_BLOCK_SIZE 512
__global__ void total(length_t * input, length_t * output, length_t len) {
    __shared__ length_t partialSum[2 * SUM_BLOCK_SIZE];
    //Load a segment of the input vector into shared memory
    length_t tid = threadIdx.x, start = 2 * blockIdx.x * SUM_BLOCK_SIZE;
    if (start + tid < len)
       partialSum[tid] = input[start + tid];
    else
       partialSum[tid] = 0;

    if (start + SUM_BLOCK_SIZE + tid < len)
       partialSum[SUM_BLOCK_SIZE + tid] = input[start + SUM_BLOCK_SIZE + tid];
    else
       partialSum[SUM_BLOCK_SIZE + tid] = 0;

    //Traverse the reduction tree
    for (int stride = SUM_BLOCK_SIZE; stride >= 1; stride >>= 1) {
       __syncthreads();
       if (tid < stride)
          partialSum[tid] += partialSum[tid+stride];
    }
    //Write the computed sum of the block to the output vector at the correct index
    if (tid == 0)
       output[blockIdx.x] = partialSum[0];
}


length_t cuStinger::sumDeviceArray(length_t* arr){
	length_t numOutputElements = nv / (SUM_BLOCK_SIZE<<1);
    if (nv % (SUM_BLOCK_SIZE<<1)) {
        numOutputElements++;
    }

	length_t* d_out = (length_t*)allocDeviceArray(nv, sizeof(length_t*));

	total<<<numOutputElements,SUM_BLOCK_SIZE>>>(d_utilized,d_out,nv);

	length_t* h_out = (int32_t*)allocHostArray(nv, sizeof(length_t*));
	
	length_t sum=0;
	copyArrayDeviceToHost(d_out, h_out, nv, sizeof(length_t));
	for(int i=0; i<numOutputElements; i++){
		 // cout << h_out[i] << ", ";
		sum+=h_out[i];
	}
	freeHostArray(h_out);
	freeDeviceArray(d_out);	
	return sum;
}



__global__ void deviceCopyMultipleAdjacencies(cuStinger* custing, vertexId_t** d_newadj, 
	vertexId_t* requireUpdates, length_t requireCount ,length_t verticesPerThreadBlock)
{
	int32_t** d_cuadj = custing->d_adj;
	length_t* d_utilized = custing->d_utilized;

	length_t v_init=blockIdx.x*verticesPerThreadBlock;
	for (int v_hat=0; v_hat<verticesPerThreadBlock; v_hat++){
		if((v_init+v_hat)>=requireCount)
			break;
		vertexId_t v=requireUpdates[v_init+v_hat];

		for(length_t e=threadIdx.x; e<d_utilized[v]; e+=blockDim.x){
			d_newadj[v][e] = d_cuadj[v][e];
		}
	}
}

void cuStinger::copyMultipleAdjacencies(vertexId_t** d_newadj, 
	vertexId_t* requireUpdates, length_t requireCount){

	dim3 numBlocks(1, 1);
	int32_t threads=32;
	dim3 threadsPerBlock(threads, 1);

	numBlocks.x = ceil((float)requireCount);
	if (numBlocks.x>16000){
		numBlocks.x=16000;
	}	
	int32_t verticesPerThreadBlock;
	if(numBlocks.x == requireCount)
		verticesPerThreadBlock=1;
	else
		verticesPerThreadBlock = ceil(float(requireCount)/float(numBlocks.x-1));

	cout << "### " << requireCount << " , " <<  numBlocks.x << " , " << verticesPerThreadBlock << " ###"  << endl; 

	deviceCopyMultipleAdjacencies<<<numBlocks,threadsPerBlock>>>(d_cuStinger,
		d_newadj, requireUpdates, requireCount, verticesPerThreadBlock);
	checkLastCudaError("Error in the first update sweep");
}



