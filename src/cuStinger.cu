#include "hip/hip_runtime.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "main.hpp"

using namespace std;



// void initializeCuStinger(cuStingerConfig);

// __global__ void devInitVertexData(cuStinger* custing,uint8_t* temp)
__global__ void devInitVertexData(cuStinger::cusVertexData *dVD,vertexId_t nv,uint8_t* temp)
{
	if(threadIdx.x!=0 || blockIdx.x!=0)
		DEV_CUSTINGER_ERROR("Number of threads and thread blocks for initializing vertex should always be one");
	// cuStinger::cusVertexData *dVD = custing->dVD;

	dVD->mem = temp;
	int32_t pos=0;
	// int32_t nv = custing->nv;

	dVD->adj 		= (cuStinger::cusEdgeData**)(dVD->getMem() + pos); 	pos+=sizeof(cuStinger::cusEdgeData*)*nv;
	dVD->edMem 		= (uint8_t**)(dVD->getMem() + pos); 				pos+=sizeof(uint8_t*)*nv;
	dVD->used 		= (length_t*)(dVD->getMem() + pos); 				pos+=sizeof(length_t)*nv;
	dVD->max        = (length_t*)(dVD->getMem() + pos); 				pos+=sizeof(length_t)*nv;
	dVD->vw         = (vweight_t*)(dVD->getMem() + pos); 				pos+=sizeof(vweight_t)*nv;
	dVD->vt         = (vtype_t*)(dVD->getMem() + pos); 					pos+=sizeof(vtype_t)*nv;

	// printf("%p\n", dVD->adj);
	// printf("%p\n", dVD->edMem);
	// printf("%p\n", dVD->used);
	// printf("%p\n", dVD->max);
	// printf("%p\n", dVD->vw);
	// printf("%p\n", dVD->vt);

}

void cuStinger::initVertexDataPointers(cuStinger::cusVertexData *dVD, uint8_t* temp){
	// devInitVertexData<<<1,1>>>(	d_cuStinger,temp);
	devInitVertexData<<<1,1>>>(	dVD,nv,temp);
}

__global__ void devInitEdgeData(cuStinger* custing, int verticesPerThreadBlock){
	vertexId_t v_init=blockIdx.x*verticesPerThreadBlock+threadIdx.x;
	length_t nv = custing->getMaxNV();
	for (vertexId_t v_hat=0; v_hat<verticesPerThreadBlock; v_hat+=blockDim.x){
		vertexId_t v=v_init+v_hat;
		if(v>=nv)
			break;
		//epv = edge per vertex
		length_t epv = custing->dVD->getMax()[v];
		int32_t pos=0;
		cuStinger::cusEdgeData *dED = custing->dVD->adj[v];

		dED->mem = custing->dVD->edMem[v];
		dED->dst = (vertexId_t*)(dED->getMem() + pos); 	pos+=sizeof(vertexId_t)*epv;
		dED->ew  = (eweight_t*)(dED->getMem() + pos); 	pos+=sizeof(eweight_t)*epv;
		dED->et  = (etype_t*)(dED->getMem() + pos); 	pos+=sizeof(etype_t)*epv;
		dED->t1  = (timestamp_t*)(dED->getMem() + pos); pos+=sizeof(timestamp_t)*epv;
		dED->t2  = (timestamp_t*)(dED->getMem() + pos); pos+=sizeof(timestamp_t)*epv;
	}
}

void cuStinger::initEdgeDataPointers(){
	dim3 numBlocks(1, 1);
	int32_t threads=64;
	dim3 threadsPerBlock(threads, 1);

	numBlocks.x = ceil((float)nv/(float)threads);
	if (numBlocks.x>16000){
		numBlocks.x=16000;
	}	

	int32_t verticesPerThreadBlock = threads;
	ceil(float(nv)/float(numBlocks.x));
	// if(numBlocks.x>1)
	// 	 verticesPerThreadBlock = ceil(float(nv)/float(numBlocks.x-1));		
	cout << "**** Number of vertices per block " << verticesPerThreadBlock << endl;
	devInitEdgeData<<<numBlocks,threadsPerBlock>>>(	d_cuStinger,verticesPerThreadBlock);
}

__global__ void devMakeGPUStinger(int32_t* d_off, int32_t* d_adj,
	int verticesPerThreadBlock,cuStinger* custing){
	length_t* d_utilized = custing->dVD->getUsed();

	int32_t v_init=blockIdx.x*verticesPerThreadBlock;
	for (int v_hat=0; v_hat<verticesPerThreadBlock; v_hat++){
		int32_t v=v_init+v_hat;
		if(v>=custing->nv)
			break;
		cuStinger::cusEdgeData* adjv = custing->dVD->adj[v];
		
		for(int32_t e=threadIdx.x; e<d_utilized[v]; e+=blockDim.x){
			// d_cuadj[v][e]=d_adj[d_off[v]+e];
			adjv->dst[e]=d_adj[d_off[v]+e];
			// adj->dst[0]=1;
		}
	}
}


void cuStinger::internalCSRTocuStinger(length_t* h_off, vertexId_t* h_adj, int ne){
	length_t* d_off = (length_t*)allocDeviceArray(nv+1,sizeof(int32_t));
	vertexId_t* d_adj = (length_t*)allocDeviceArray(ne,sizeof(int32_t));
	copyArrayHostToDevice(h_off,d_off,nv,sizeof(length_t));
	copyArrayHostToDevice(h_adj,d_adj,ne,sizeof(vertexId_t));

	dim3 numBlocks(1, 1);
	int32_t threads=64;
	dim3 threadsPerBlock(threads, 1);

	numBlocks.x = ceil((float)nv/(float)threads);
	if (numBlocks.x>16000){
		numBlocks.x=16000;
	}	

	int32_t verticesPerThreadBlock = ceil(float(nv)/float(numBlocks.x-1));

	devMakeGPUStinger<<<numBlocks,threadsPerBlock>>>(d_off,d_adj,verticesPerThreadBlock, d_cuStinger);

	freeDeviceArray(d_adj);	
	freeDeviceArray(d_off);
}



#define SUM_BLOCK_SIZE 512
__global__ void devSumArray(length_t * input, length_t * output, length_t len) {
    __shared__ length_t partialSum[2 * SUM_BLOCK_SIZE];
    //Load a segment of the input vector into shared memory
    length_t tid = threadIdx.x, start = 2 * blockIdx.x * SUM_BLOCK_SIZE;
    if (start + tid < len)
       partialSum[tid] = input[start + tid];
    else
       partialSum[tid] = 0;

    if (start + SUM_BLOCK_SIZE + tid < len)
       partialSum[SUM_BLOCK_SIZE + tid] = input[start + SUM_BLOCK_SIZE + tid];
    else
       partialSum[SUM_BLOCK_SIZE + tid] = 0;

    //Traverse the reduction tree
    for (int stride = SUM_BLOCK_SIZE; stride >= 1; stride >>= 1) {
       __syncthreads();
       if (tid < stride)
          partialSum[tid] += partialSum[tid+stride];
    }
    //Write the computed sum of the block to the output vector at the correct index
    if (tid == 0)
       output[blockIdx.x] = partialSum[0];
}

length_t cuStinger::sumDeviceArray(length_t* arr, length_t len){
	length_t numOutputElements = len / (SUM_BLOCK_SIZE<<1);
    if (len % (SUM_BLOCK_SIZE<<1)) {
        numOutputElements++;
    }

	length_t* d_out = (length_t*)allocDeviceArray(len, sizeof(length_t*));

	devSumArray<<<numOutputElements,SUM_BLOCK_SIZE>>>(arr,d_out,len);

	length_t* h_out = (int32_t*)allocHostArray(len, sizeof(length_t*));
	
	length_t sum=0;
	copyArrayDeviceToHost(d_out, h_out, len, sizeof(length_t));
	for(int i=0; i<numOutputElements; i++){
		sum+=h_out[i];
	}
	freeHostArray(h_out);
	freeDeviceArray(d_out);	
	return sum;
}

__global__ void deviceCopyMultipleAdjacencies(cuStinger* custing, cuStinger::cusVertexData* olddVD, 
	vertexId_t* requireUpdates, length_t requireCount ,length_t verticesPerThreadBlock)
{
	// int32_t** d_cuadj = custing->d_adj;
	// length_t* d_utilized = custing->getDeviceUsed();

	length_t v_init=blockIdx.x*verticesPerThreadBlock;
	for (int v_hat=0; v_hat<verticesPerThreadBlock; v_hat++){
		if((v_init+v_hat)>=requireCount)
			break;
		vertexId_t v=requireUpdates[v_init+v_hat];

		cuStinger::cusEdgeData *dED = custing->dVD->adj[v];
		cuStinger::cusEdgeData *olddED = olddVD->adj[v];

		//epv = edge per vertex
		length_t epv = olddVD->getMax()[v];
		int32_t pos=0;

		dED->mem = custing->dVD->edMem[v];
		dED->dst = (vertexId_t*)(dED->getMem() + pos); 	pos+=sizeof(vertexId_t)*epv;
		dED->ew  = (eweight_t*)(dED->getMem() + pos); 	pos+=sizeof(eweight_t)*epv;
		dED->et  = (etype_t*)(dED->getMem() + pos); 	pos+=sizeof(etype_t)*epv;
		dED->t1  = (timestamp_t*)(dED->getMem() + pos); pos+=sizeof(timestamp_t)*epv;
		dED->t2  = (timestamp_t*)(dED->getMem() + pos); pos+=sizeof(timestamp_t)*epv;

		for(length_t e=threadIdx.x; e<olddVD->getUsed()[v]; e+=blockDim.x){
			dED->dst[e] = olddED->dst[e];	

		}
	}
}


void cuStinger::copyMultipleAdjacencies(cusVertexData* olddVD, 
	vertexId_t* requireUpdates, length_t requireCount){

	dim3 numBlocks(1, 1);
	int32_t threads=32;
	dim3 threadsPerBlock(threads, 1);

	numBlocks.x = ceil((float)requireCount);
	if (numBlocks.x>16000){
		numBlocks.x=16000;
	}	
	int32_t verticesPerThreadBlock;
	if(numBlocks.x == requireCount)
		verticesPerThreadBlock=1;
	else
		verticesPerThreadBlock = ceil(float(requireCount)/float(numBlocks.x-1));

	cout << "### " << requireCount << " , " <<  numBlocks.x << " , " << verticesPerThreadBlock << " ###"  << endl; 

	deviceCopyMultipleAdjacencies<<<numBlocks,threadsPerBlock>>>(d_cuStinger,
		olddVD, requireUpdates, requireCount, verticesPerThreadBlock);
	checkLastCudaError("Error in the first update sweep");
}



