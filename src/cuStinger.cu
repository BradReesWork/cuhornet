#include "hip/hip_runtime.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "main.hpp"

using namespace std;



// void initializeCuStinger(cuStingerConfig);

__global__ void devInitVertexData(cuStinger* custing,uint8_t* temp)
{
	if(threadIdx.x!=0 || blockIdx.x!=0)
		DEV_CUSTINGER_ERROR("Number of threads and thread blocks for initializing vertex should always be one");
	cuStinger::cusVertexData *dVD = custing->dVD;

	dVD->mem = temp;
	int32_t pos=0;
	int32_t nv = custing->nv;

	// printf("Vertex: From the device : %p \n",dVD);
	// printf("Vertex: From the device : %p \n",temp);

	dVD->adj 		= (cuStinger::cusEdgeData**)(dVD->getMem() + pos); 	pos+=sizeof(cuStinger::cusEdgeData*)*nv;
	dVD->edMem 		= (uint8_t**)(dVD->getMem() + pos); 				pos+=sizeof(uint8_t*)*nv;
	dVD->used 		= (length_t*)(dVD->getMem() + pos); 				pos+=sizeof(length_t)*nv;
	dVD->max        = (length_t*)(dVD->getMem() + pos); 				pos+=sizeof(length_t)*nv;
	dVD->vw         = (vweight_t*)(dVD->getMem() + pos); 				pos+=sizeof(vweight_t)*nv;
	dVD->vt         = (vtype_t*)(dVD->getMem() + pos); 					pos+=sizeof(vtype_t)*nv;
}

void cuStinger::initVertexDataPointers(uint8_t* temp){
	devInitVertexData<<<1,1>>>(	d_cuStinger,temp);
}

__global__ void devInitEdgeData(cuStinger* custing, int verticesPerThreadBlock)
{
	// if(threadIdx.x==0 && blockIdx.x==10)
	// 	printf("The number of vertices is : %d\n", custing->nv);

	vertexId_t v_init=blockIdx.x*verticesPerThreadBlock+threadIdx.x;
	length_t nv = custing->nv;
	for (vertexId_t v_hat=0; v_hat<verticesPerThreadBlock; v_hat+=blockDim.x){
		vertexId_t v=v_init+v_hat;
		if(v>=custing->nv)
			break;
		//epv = edge per vertex
		length_t epv = custing->getDeviceMax()[v];
		int32_t pos=0;
		cuStinger::cusEdgeData *dED = custing->dVD->adj[v];

		dED->mem = custing->dVD->edMem[v];

		if(threadIdx.x==0 && blockIdx.x==10) printf("####%d,",epv);
		dED->dst = (vertexId_t*)(dED->getMem() + pos); 	pos+=sizeof(vertexId_t)*epv;
		if(threadIdx.x==0 && blockIdx.x==10) printf("%p,",dED->dst);
		dED->ew  = (eweight_t*)(dED->getMem() + pos); 	pos+=sizeof(eweight_t)*epv;
		if(threadIdx.x==0 && blockIdx.x==10) printf("%d,",pos);
		dED->et  = (etype_t*)(dED->getMem() + pos); 	pos+=sizeof(etype_t)*epv;
		if(threadIdx.x==0 && blockIdx.x==10) printf("%d,",pos);
		dED->t1  = (timestamp_t*)(dED->getMem() + pos); pos+=sizeof(timestamp_t)*epv;
		if(threadIdx.x==0 && blockIdx.x==10) printf("%d,",pos);
		dED->t2  = (timestamp_t*)(dED->getMem() + pos); pos+=sizeof(timestamp_t)*epv;
		if(threadIdx.x==0 && blockIdx.x==10) printf("%d,",pos);
		if(threadIdx.x==0 && blockIdx.x==10) printf("####\n");
	}
}

void cuStinger::initEdgeDataPointers(){
	dim3 numBlocks(1, 1);
	int32_t threads=64;
	dim3 threadsPerBlock(threads, 1);

	numBlocks.x = ceil((float)nv/(float)threads);
	if (numBlocks.x>16000){
		numBlocks.x=16000;
	}	

	int32_t verticesPerThreadBlock = threads;
	ceil(float(nv)/float(numBlocks.x));
	// if(numBlocks.x>1)
	// 	 verticesPerThreadBlock = ceil(float(nv)/float(numBlocks.x-1));		
	cout << "**** Number of vertices per block " << verticesPerThreadBlock << endl;
	devInitEdgeData<<<numBlocks,threadsPerBlock>>>(	d_cuStinger,verticesPerThreadBlock);
}

__global__ void devMakeGPUStinger(int32_t* d_off, int32_t* d_adj,
	int verticesPerThreadBlock,cuStinger* custing){
	length_t* d_utilized = custing->getDeviceUsed();

	int32_t v_init=blockIdx.x*verticesPerThreadBlock;
	for (int v_hat=0; v_hat<verticesPerThreadBlock; v_hat++){
		int32_t v=v_init+v_hat;
		if(v>=custing->nv)
			break;
		cuStinger::cusEdgeData* adjv = custing->dVD->adj[v];

		for(int32_t e=threadIdx.x; e<d_utilized[v]; e+=blockDim.x){
			// d_cuadj[v][e]=d_adj[d_off[v]+e];
			adjv->dst[e]=d_adj[d_off[v]+e];
			// adj->dst[0]=1;
		}
	}
}


void cuStinger::internalCSRTocuStinger(length_t* h_off, vertexId_t* h_adj, int ne){
	length_t* d_off = (length_t*)allocDeviceArray(nv+1,sizeof(int32_t));
	vertexId_t* d_adj = (length_t*)allocDeviceArray(ne,sizeof(int32_t));
	copyArrayHostToDevice(h_off,d_off,nv,sizeof(length_t));
	copyArrayHostToDevice(h_adj,d_adj,ne,sizeof(vertexId_t));

	dim3 numBlocks(1, 1);
	int32_t threads=64;
	dim3 threadsPerBlock(threads, 1);

	numBlocks.x = ceil((float)nv/(float)threads);
	if (numBlocks.x>16000){
		numBlocks.x=16000;
	}	

	int32_t verticesPerThreadBlock = ceil(float(nv)/float(numBlocks.x-1));

	devMakeGPUStinger<<<numBlocks,threadsPerBlock>>>(d_off,d_adj,verticesPerThreadBlock, d_cuStinger);

	freeDeviceArray(d_adj);	
	freeDeviceArray(d_off);
}



#define SUM_BLOCK_SIZE 512
__global__ void total(length_t * input, length_t * output, length_t len) {
    __shared__ length_t partialSum[2 * SUM_BLOCK_SIZE];
    //Load a segment of the input vector into shared memory
    length_t tid = threadIdx.x, start = 2 * blockIdx.x * SUM_BLOCK_SIZE;
    if (start + tid < len)
       partialSum[tid] = input[start + tid];
    else
       partialSum[tid] = 0;

    if (start + SUM_BLOCK_SIZE + tid < len)
       partialSum[SUM_BLOCK_SIZE + tid] = input[start + SUM_BLOCK_SIZE + tid];
    else
       partialSum[SUM_BLOCK_SIZE + tid] = 0;

    //Traverse the reduction tree
    for (int stride = SUM_BLOCK_SIZE; stride >= 1; stride >>= 1) {
       __syncthreads();
       if (tid < stride)
          partialSum[tid] += partialSum[tid+stride];
    }
    //Write the computed sum of the block to the output vector at the correct index
    if (tid == 0)
       output[blockIdx.x] = partialSum[0];
}


length_t cuStinger::sumDeviceArray(length_t* arr, length_t len){
	length_t numOutputElements = len / (SUM_BLOCK_SIZE<<1);
    if (len % (SUM_BLOCK_SIZE<<1)) {
        numOutputElements++;
    }

	length_t* d_out = (length_t*)allocDeviceArray(len, sizeof(length_t*));

	total<<<numOutputElements,SUM_BLOCK_SIZE>>>(arr,d_out,len);

	length_t* h_out = (int32_t*)allocHostArray(len, sizeof(length_t*));
	
	length_t sum=0;
	copyArrayDeviceToHost(d_out, h_out, len, sizeof(length_t));
	for(int i=0; i<numOutputElements; i++){
		 // cout << h_out[i] << ", ";
		sum+=h_out[i];
	}
	freeHostArray(h_out);
	freeDeviceArray(d_out);	
	return sum;
}



__global__ void deviceCopyMultipleAdjacencies(cuStinger* custing, vertexId_t** d_newadj, 
	vertexId_t* requireUpdates, length_t requireCount ,length_t verticesPerThreadBlock)
{
	// int32_t** d_cuadj = custing->d_adj;
	length_t* d_utilized = custing->getDeviceUsed();

	length_t v_init=blockIdx.x*verticesPerThreadBlock;
	for (int v_hat=0; v_hat<verticesPerThreadBlock; v_hat++){
		if((v_init+v_hat)>=requireCount)
			break;
		vertexId_t v=requireUpdates[v_init+v_hat];

		for(length_t e=threadIdx.x; e<d_utilized[v]; e+=blockDim.x){
			// d_newadj[v][e] = d_cuadj[v][e];
		}
	}
}

void cuStinger::copyMultipleAdjacencies(vertexId_t** d_newadj, 
	vertexId_t* requireUpdates, length_t requireCount){

	dim3 numBlocks(1, 1);
	int32_t threads=32;
	dim3 threadsPerBlock(threads, 1);

	numBlocks.x = ceil((float)requireCount);
	if (numBlocks.x>16000){
		numBlocks.x=16000;
	}	
	int32_t verticesPerThreadBlock;
	if(numBlocks.x == requireCount)
		verticesPerThreadBlock=1;
	else
		verticesPerThreadBlock = ceil(float(requireCount)/float(numBlocks.x-1));

	cout << "### " << requireCount << " , " <<  numBlocks.x << " , " << verticesPerThreadBlock << " ###"  << endl; 

	deviceCopyMultipleAdjacencies<<<numBlocks,threadsPerBlock>>>(d_cuStinger,
		d_newadj, requireUpdates, requireCount, verticesPerThreadBlock);
	checkLastCudaError("Error in the first update sweep");
}



