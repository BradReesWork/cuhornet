#include "hip/hip_runtime.h"
/**
 * @author Federico Busato                                                  <br>
 *         Univerity of Verona, Dept. of Computer Science                   <br>
 *         federico.busato@univr.it
 * @date April, 2017
 * @version v2
 *
 * @copyright Copyright © 2017 cuStinger. All rights reserved.
 *
 * @license{<blockquote>
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 * * Neither the name of the copyright holder nor the names of its
 *   contributors may be used to endorse or promote products derived from
 *   this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 * </blockquote>}
 */
#include "Core/cuStinger.hpp"
#include "GlobalSpace.cuh"                //d_nV
#include "Core/cuStingerTypes.cuh"        //VertexBasicData
#include "Support/Device/CubWrapper.cuh"  //CubSortByValue

namespace custinger {

void cuStinger::initializeVertexGlobal(byte_t* (&vertex_data_ptrs)[NUM_VTYPES])
                                       noexcept {
    cuMemcpyToSymbol(_nV, d_nV);
    cuMemcpyToSymbol(vertex_data_ptrs, NUM_VTYPES, d_vertex_data_ptrs);
}

//==============================================================================

__device__ int d_array[10];

__global__ void printKernel() {
    for (vid_t i = 0; i < d_nV; i++) {
        auto vertex = Vertex(i);
        auto degree = vertex.degree();
        //auto field0 = vertex.field<0>();
        printf("%d [%d, %d]:    ", i, vertex.degree(), vertex.limit());

        for (degree_t j = 0; j < vertex.degree(); j++) {
            auto   edge = vertex.edge(j);
            /*auto weight = edge.weight();
            auto  time1 = edge.time_stamp1();
            auto field0 = edge.field<0>();
            auto field1 = edge.field<1>();*/

            printf("%d    ", edge.dst());
        //    d_array[j] = edge.dst();
        }
        printf("\n");
    }
    //printf("\n");
    //from RAW:
    //
    //for (vid_t i = 0; i < d_nV; i++) {
    //  for (degree_t j = 0; j < vertex.degrees(); j++) {
    //       auto edge = vertex.edge(i);
    //----------------------------------------------------
    //to PROPOSED:
    //
    //for (auto v : VertexSet) {
    //  for (auto edge : v) {
}

void cuStinger::print() noexcept {
    if (sizeof(degree_t) == 4 && sizeof(vid_t) == 4) {
        printKernel<<<1, 1>>>();
        CHECK_CUDA_ERROR
    }
    else {
        WARNING("Graph print is enabled only with degree_t/vid_t of size"
                " 4 bytes")
    }
}

void cuStinger::transpose() noexcept {
    mem_manager.clear();

    eoff_t* d_csr_offsets, *d_counts_out;
    vid_t*  d_csr_edges, *d_unique_out, *d_csr_edges_sorted;
    cuMalloc(d_csr_offsets, _nV + 1);
    cuMalloc(d_csr_edges, _nE);
    cuMalloc(d_csr_edges_sorted, _nE);
    cuMalloc(d_counts_out, _nV + 1);
    cuMalloc(d_unique_out, _nV);
    cuMemcpyToDeviceAsync(_csr_offsets, _nV + 1, d_csr_offsets);
    cuMemcpyToDeviceAsync(_csr_edges, _nE, d_csr_edges);
    cuMemcpyToDeviceAsync(0, d_counts_out + _nV);

    xlib::CubSortByValue<vid_t>(d_csr_edges, _nE, d_csr_edges_sorted, _nV - 1);
    xlib::CubRunLengthEncode<vid_t, eoff_t>(d_csr_edges_sorted, _nE,
                                            d_unique_out, d_counts_out);
    cuMemset0x00(d_unique_out, _nV);
    xlib::CubExclusiveSum<eoff_t>(d_counts_out, _nV + 1);

    //transpose_edges(d_csr_offsets, d_csr_edges, d_counts_out, d_unique_out
    //               [](atomicAdd(&, 1));
    cuFree(d_csr_offsets, d_csr_edges, d_csr_edges_sorted, d_counts_out,
           d_unique_out);

    _csr_offsets = new eoff_t[_nV + 1];
    _csr_edges   = new vid_t[_nV + 1];
    cuMemcpyToHostAsync(d_csr_offsets, _nV + 1,
                        const_cast<eoff_t*>(_csr_offsets));
    cuMemcpyToHostAsync(d_csr_edges, _nE, const_cast<vid_t*>(_csr_edges));
    _internal_csr_data = true;
    initialize();
}

} // namespace custinger
