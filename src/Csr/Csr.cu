#include "hip/hip_runtime.h"
/**
 * @author Federico Busato                                                  <br>
 *         Univerity of Verona, Dept. of Computer Science                   <br>
 *         federico.busato@univr.it
 * @date April, 2017
 * @version v2
 *
 * @copyright Copyright © 2017 cuStinger. All rights reserved.
 *
 * @license{<blockquote>
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 * * Neither the name of the copyright holder nor the names of its
 *   contributors may be used to endorse or promote products derived from
 *   this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 * </blockquote>}
 */
#include "Csr/Csr.hpp"
#include "GlobalSpace.cuh"          //d_nV
#include "Csr/CsrTypes.cuh"

namespace csr {

void Csr::initializeGlobal(byte_t* (&vertex_data_ptrs)[NUM_VTYPES],
                           byte_t* (&edge_data_ptrs)[NUM_ETYPES]) noexcept {
    cuMemcpyToSymbol(_nV, d_nV);
    cuMemcpyToSymbol(vertex_data_ptrs, NUM_VTYPES, d_vertex_data_ptrs);
    cuMemcpyToSymbol(edge_data_ptrs, NUM_VTYPES, d_edge_data_ptrs);
}

//==============================================================================

__global__ void printKernel() {
    for (vid_t i = 0; i < d_nV; i++) {
        auto vertex = Vertex(i);
        auto degree = vertex.degree();
        //auto field0 = vertex.field<0>();
        printf("%d [%d]:    ", i, vertex.degree());

        for (degree_t j = 0; j < vertex.degree(); j++) {
            auto   edge = vertex.edge(j);
            /*auto weight = edge.weight();
            auto  time1 = edge.time_stamp1();
            auto field0 = edge.field<0>();
            auto field1 = edge.field<1>();*/

            printf("%d    ", edge.dst());
        //    d_array[j] = edge.dst();
        }
        printf("\n");
    }
}

void Csr::print() noexcept {
    if (sizeof(degree_t) == 4 && sizeof(vid_t) == 4) {
        printKernel<<<1, 1>>>();
        CHECK_CUDA_ERROR
    }
    else
        WARNING("Graph print is enable only with degree_t/vid_t of size 4 bytes")
}

} // namespace csr
