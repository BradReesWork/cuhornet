#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <inttypes.h>

#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#include "cct.hpp"

__device__ void conditionalWarpReduce(volatile triangle_t* sharedData,int blockSize,int dataLength){
  if(blockSize >= dataLength){
    if(threadIdx.x < (dataLength/2))
    {sharedData[threadIdx.x] += sharedData[threadIdx.x+(dataLength/2)];}
    __syncthreads();
  }
}

__device__ void warpReduce(triangle_t* __restrict__ outDataPtr,
    volatile triangle_t* __restrict__ sharedData,int blockSize){
  conditionalWarpReduce(sharedData,blockSize,64);
  conditionalWarpReduce(sharedData,blockSize,32);
  conditionalWarpReduce(sharedData,blockSize,16);
  conditionalWarpReduce(sharedData,blockSize,8);
  conditionalWarpReduce(sharedData,blockSize,4);
  if(threadIdx.x == 0)
    {*outDataPtr= sharedData[0] + sharedData[1];}
  __syncthreads();
}

__device__ void conditionalReduce(volatile triangle_t* __restrict__ sharedData,int blockSize,int dataLength){
	if(blockSize >= dataLength){
		if(threadIdx.x < (dataLength/2))
		{sharedData[threadIdx.x] += sharedData[threadIdx.x+(dataLength/2)];}
		__syncthreads();
	}
	if((blockSize < dataLength) && (blockSize > (dataLength/2))){
		if(threadIdx.x+(dataLength/2) < blockSize){
			sharedData[threadIdx.x] += sharedData[threadIdx.x+(dataLength/2)];
		}
		__syncthreads();
	}
}

__device__ void blockReduce(triangle_t* __restrict__ outGlobalDataPtr,
    volatile triangle_t* __restrict__ sharedData,int blockSize){
  __syncthreads();
  conditionalReduce(sharedData,blockSize,1024);
  conditionalReduce(sharedData,blockSize,512);
  conditionalReduce(sharedData,blockSize,256);
  conditionalReduce(sharedData,blockSize,128);

  warpReduce(outGlobalDataPtr, sharedData, blockSize);
  __syncthreads();
}

__device__ void initialize(const vertexId_t diag_id, const length_t u_len, length_t v_len,
    length_t* const __restrict__ u_min, length_t* const __restrict__ u_max,
    length_t* const __restrict__ v_min, length_t* const __restrict__ v_max,
    int* const __restrict__ found)
{
	if (diag_id == 0){
		*u_min=*u_max=*v_min=*v_max=0;
		*found=1;
	}
	else if (diag_id < u_len){
		*u_min=0; *u_max=diag_id;
		*v_max=diag_id;*v_min=0;
	}
	else if (diag_id < v_len){
		*u_min=0; *u_max=u_len;
		*v_max=diag_id;*v_min=diag_id-u_len;
	}
	else{
		*u_min=diag_id-v_len; *u_max=u_len;
		*v_min=diag_id-u_len; *v_max=v_len;
	}
}

__device__ void workPerThread(const length_t uLength, const length_t vLength, 
	const int threadsPerIntersection, const int threadId,
    int * const __restrict__ outWorkPerThread, int * const __restrict__ outDiagonalId){
  int totalWork = uLength + vLength;
  int remainderWork = totalWork%threadsPerIntersection;
  int workPerThread = totalWork/threadsPerIntersection;

  int longDiagonals  = (threadId > remainderWork) ? remainderWork:threadId;
  int shortDiagonals = (threadId > remainderWork) ? (threadId - remainderWork):0;

  *outDiagonalId = ((workPerThread+1)*longDiagonals) + (workPerThread*shortDiagonals);
  *outWorkPerThread = workPerThread + (threadId < remainderWork);
}

__device__ void bSearch(unsigned int found, const vertexId_t diagonalId,
    vertexId_t const * const __restrict__ uNodes, vertexId_t const * const __restrict__ vNodes,
    length_t const * const __restrict__ uLength, 
    length_t * const __restrict__ outUMin, length_t * const __restrict__ outUMax,
    length_t * const __restrict__ outVMin, length_t * const __restrict__ outVMax,    
    length_t * const __restrict__ outUCurr,
    length_t * const __restrict__ outVCurr){
  	length_t length;
	
	while(!found) {
	    *outUCurr = (*outUMin + *outUMax)>>1;
	    *outVCurr = diagonalId - *outUCurr;
	    if(*outVCurr >= *outVMax){
			length = *outUMax - *outUMin;
			if(length == 1){
				found = 1;
				continue;
			}
	    }

	    unsigned int comp1 = uNodes[*outUCurr] > vNodes[*outVCurr-1];
	    unsigned int comp2 = uNodes[*outUCurr-1] > vNodes[*outVCurr];
	    if(comp1 && !comp2){
			found = 1;
	    }
	    else if(comp1){
	      *outVMin = *outVCurr;
	      *outUMax = *outUCurr;
	    }
	    else{
	      *outVMax = *outVCurr;
	      *outUMin = *outUCurr;
	    }
  	}

	if((*outVCurr >= *outVMax) && (length == 1) && (*outVCurr > 0) &&
	(*outUCurr > 0) && (*outUCurr < (*uLength - 1))){
		unsigned int comp1 = uNodes[*outUCurr] > vNodes[*outVCurr - 1];
		unsigned int comp2 = uNodes[*outUCurr - 1] > vNodes[*outVCurr];
		if(!comp1 && !comp2){(*outUCurr)++; (*outVCurr)--;}
	}
}

__device__ int fixStartPoint(const length_t uLength, const length_t vLength,
    length_t * const __restrict__ uCurr, length_t * const __restrict__ vCurr,
    vertexId_t const * const __restrict__ uNodes, vertexId_t const * const __restrict__ vNodes){
	
	unsigned int uBigger = (*uCurr > 0) && (*vCurr < vLength) && (uNodes[*uCurr-1] == vNodes[*vCurr]);
	unsigned int vBigger = (*vCurr > 0) && (*uCurr < uLength) && (vNodes[*vCurr-1] == uNodes[*uCurr]);
	*uCurr += vBigger;
	*vCurr += uBigger;
	return (uBigger + vBigger);
}

__device__ void intersectCount(const length_t uLength, const length_t vLength,
    vertexId_t const * const __restrict__ uNodes, vertexId_t const * const __restrict__ vNodes,
    length_t * const __restrict__ uCurr, length_t * const __restrict__ vCurr,
    int * const __restrict__ workIndex, int * const __restrict__ workPerThread,
    int * const __restrict__ triangles, int found)
{
  if((*uCurr < uLength) && (*vCurr < vLength)){
    int comp;
    while(*workIndex < *workPerThread){
		comp = uNodes[*uCurr] - vNodes[*vCurr];
		*triangles += (comp == 0);
		*uCurr += (comp <= 0);
		*vCurr += (comp >= 0);
		*workIndex += (comp == 0) + 1;

		if((*vCurr == vLength) || (*uCurr == uLength)){
			break;
		}
    }
    *triangles -= ((comp == 0) && (*workIndex > *workPerThread) && (found));
  }
}


// u_len < v_len
__device__ triangle_t count_triangles(vertexId_t u, vertexId_t const * const __restrict__ u_nodes, length_t u_len,
    vertexId_t v, vertexId_t const * const __restrict__ v_nodes, length_t v_len, int threads_per_block,
    volatile vertexId_t* __restrict__ firstFound, int tId)
{
	// Partitioning the work to the multiple thread of a single GPU processor. The threads should get a near equal number of the elements to Tersect - this number will be off by 1.
	int work_per_thread, diag_id;
	workPerThread(u_len, v_len, threads_per_block, tId, &work_per_thread, &diag_id);
	triangle_t triangles = 0;
	int work_index = 0,found=0;
	length_t u_min,u_max,v_min,v_max,u_curr,v_curr;

	firstFound[tId]=0;

	if(work_per_thread>0){
		// For the binary search, we are figuring out the initial poT of search.
		initialize(diag_id, u_len, v_len,&u_min, &u_max,&v_min, &v_max,&found);
    	u_curr = 0; v_curr = 0;

	    bSearch(found, diag_id, u_nodes, v_nodes, &u_len, &u_min, &u_max, &v_min,
        &v_max, &u_curr, &v_curr);

    	int sum = fixStartPoint(u_len, v_len, &u_curr, &v_curr, u_nodes, v_nodes);
    	work_index += sum;
	    if(tId > 0)
	      firstFound[tId-1] = sum;
	    triangles += sum;
	    intersectCount(u_len, v_len, u_nodes, v_nodes, &u_curr, &v_curr,
	        &work_index, &work_per_thread, &triangles, firstFound[tId]);
	}
	return triangles;
}

__device__ void workPerBlock(const length_t numVertices,
    length_t * const __restrict__ outMpStart,
    length_t * const __restrict__ outMpEnd, int blockSize)
{
	length_t verticesPerMp = numVertices/gridDim.x;
	length_t remainderBlocks = numVertices % gridDim.x;
	length_t extraVertexBlocks = (blockIdx.x > remainderBlocks)? remainderBlocks:blockIdx.x;
	length_t regularVertexBlocks = (blockIdx.x > remainderBlocks)? blockIdx.x - remainderBlocks:0;

	length_t mpStart = ((verticesPerMp+1)*extraVertexBlocks) + (verticesPerMp*regularVertexBlocks);
	*outMpStart = mpStart;
	*outMpEnd = mpStart + verticesPerMp + (blockIdx.x < remainderBlocks);
}


__global__ void devicecuStingerNewTriangles(cuStinger* custing, BatchUpdateData *bud,
    triangle_t * const __restrict__ outPutTriangles, const int threads_per_block,
    const int number_blocks, const int shifter)
{
	length_t batchSize = *(bud->getBatchSize());
	// Partitioning the work to the multiple thread of a single GPU processor. The threads should get a near equal number of the elements to intersect - this number will be off by no more than one.
	int tx = threadIdx.x;
 	length_t this_mp_start, this_mp_stop;

	const int blockSize = blockDim.x;
	workPerBlock(batchSize, &this_mp_start, &this_mp_stop, blockSize);

	__shared__ vertexId_t firstFound[1024];

	length_t adj_offset=tx>>shifter;
	length_t* firstFoundPos=firstFound + (adj_offset<<shifter);
	for (length_t edge = this_mp_start+adj_offset; edge < this_mp_stop; edge+=number_blocks){
		vertexId_t src = bud->getSrc()[edge];
		vertexId_t dest= bud->getDst()[edge];
		length_t srcLen=custing->dVD->getUsed()[src];
		length_t destLen=custing->dVD->getUsed()[dest];

		bool avoidCalc = (src == dest) || (destLen < 2) || (srcLen < 2);
		if(avoidCalc)
			continue;

		bool sourceSmaller = (srcLen<destLen);
        vertexId_t small = sourceSmaller? src : dest;
        vertexId_t large = sourceSmaller? dest : src;
        length_t small_len = sourceSmaller? srcLen : destLen;
        length_t large_len = sourceSmaller? destLen : srcLen;

        const vertexId_t* small_ptr = custing->dVD->getAdj()[small]->dst;
        const vertexId_t* large_ptr = custing->dVD->getAdj()[large]->dst;

		triangle_t tCount = count_triangles(small, small_ptr, small_len,
								large,large_ptr, large_len,
								threads_per_block,firstFoundPos,
								tx%threads_per_block);

		atomicAdd(outPutTriangles + src, tCount);
		__syncthreads();
	}
}

template <typename T>
T sumTriangleArrayTEST(T* h_triangles, vertexId_t nv){	
	T sum=0;
	for(vertexId_t sd=0; sd<(nv);sd++){
	  sum+=h_triangles[sd];
	}
	return sum;
}

__global__ void deviceBUThreeTriangles (const vertexId_t nv,
    length_t const * const __restrict__ d_off, vertexId_t const * const __restrict__ d_ind,
    triangle_t * const __restrict__ outPutTriangles, const int threads_per_block,
    const int number_blocks, const int shifter)
{
	// Partitioning the work to the multiple thread of a single GPU processor. The threads should get a near equal number of the elements to intersect - this number will be off by no more than one.
	int tx = threadIdx.x;
 	vertexId_t this_mp_start, this_mp_stop;

	const int blockSize = blockDim.x;
	workPerBlock(nv, &this_mp_start, &this_mp_stop, blockSize);

	__shared__ triangle_t  s_triangles[1024];
	__shared__ vertexId_t firstFound[1024];

	length_t adj_offset=tx>>shifter;
	length_t* firstFoundPos=firstFound + (adj_offset<<shifter);
	for (vertexId_t src = this_mp_start; src < this_mp_stop; src++){
		int srcLen=d_off[src+1]-d_off[src];
	    triangle_t tCount = 0;	    

		for(int iter=d_off[src]+adj_offset; iter<d_off[src+1]; iter+=number_blocks){
			int dest = d_ind[iter];
			int destLen = d_off[dest+1]-d_off[dest];

			bool avoidCalc = (src == dest) || (destLen < 2) || (srcLen < 2);
			if(avoidCalc)
				continue;

	        bool sourceSmaller = (srcLen<destLen);
	        vertexId_t small = sourceSmaller? src : dest;
	        vertexId_t large = sourceSmaller? dest : src;
	        length_t small_len = sourceSmaller? srcLen : destLen;
	        length_t large_len = sourceSmaller? destLen : srcLen;

	        vertexId_t const * const small_ptr = d_ind + d_off[small];
	        vertexId_t const * const large_ptr = d_ind + d_off[large];

	        tCount += count_triangles(small, small_ptr, small_len,
						large,large_ptr, large_len,
						threads_per_block,firstFoundPos,
						tx%threads_per_block);
		}
		s_triangles[tx] = tCount;
		blockReduce(&outPutTriangles[src],s_triangles,blockSize);
	}
}

__global__ void deviceBUTwoCUOneTriangles (cuStinger* custing, const vertexId_t nv,
    length_t const * const __restrict__ d_off, vertexId_t const * const __restrict__ d_ind,
    triangle_t * const __restrict__ outPutTriangles, const int threads_per_block,
    const int number_blocks, const int shifter)
{
	// Partitioning the work to the multiple thread of a single GPU processor. The threads should get a near equal number of the elements to intersect - this number will be off by no more than one.
	int tx = threadIdx.x;
 	vertexId_t this_mp_start, this_mp_stop;

	const int blockSize = blockDim.x;
	workPerBlock(nv, &this_mp_start, &this_mp_stop, blockSize);

	__shared__ triangle_t  s_triangles[1024];
	__shared__ vertexId_t firstFound[1024];

	length_t adj_offset=tx>>shifter;
	length_t* firstFoundPos=firstFound + (adj_offset<<shifter);
	for (vertexId_t src = this_mp_start; src < this_mp_stop; src++){
		int srcLen=d_off[src+1]-d_off[src];
	    triangle_t tCount = 0;	    

		for(int iter=d_off[src]+adj_offset; iter<d_off[src+1]; iter+=number_blocks){
			int dest = d_ind[iter];
			int destLen = custing->dVD->getUsed()[dest];

			bool avoidCalc = (src == dest) || (destLen < 2) || (srcLen < 2);
			if(avoidCalc)
				continue;

	        vertexId_t const * const src_ptr = d_ind + d_off[src];
	        vertexId_t const * const dst_ptr = custing->dVD->getAdj()[dest]->dst;

	        bool sourceSmaller = (srcLen<destLen);
	        vertexId_t small = sourceSmaller? src : dest;
	        vertexId_t large = sourceSmaller? dest : src;
	        length_t small_len = sourceSmaller? srcLen : destLen;
	        length_t large_len = sourceSmaller? destLen : srcLen;

	        vertexId_t const * const small_ptr = sourceSmaller? src_ptr : dst_ptr;
	        vertexId_t const * const large_ptr = sourceSmaller? dst_ptr : src_ptr;

	        tCount += count_triangles(small, small_ptr, small_len,
						large,large_ptr, large_len,
						threads_per_block,firstFoundPos,
						tx%threads_per_block);
		}
		s_triangles[tx] = tCount;
		blockReduce(&outPutTriangles[src],s_triangles,blockSize);
	}
}


__global__ void calcEdgelistLengths(BatchUpdateData *bud, length_t* const __restrict__ ell){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	length_t batchSize = *(bud->getBatchSize());
	if (tid < batchSize) {
		vertexId_t src = bud->getSrc()[tid];
		atomicAdd(ell+src, 1);
	}
}

__global__ void copyIndices(BatchUpdateData *bud, vertexId_t* const __restrict__ ind,
	vertexId_t* const __restrict__ seg,	length_t* const __restrict__ off,
	length_t* const __restrict__ ell){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	length_t batchSize = *(bud->getBatchSize());
	if (tid < batchSize)
	{
		vertexId_t src = bud->getSrc()[tid];
		// Start filling up from the end of the edge list like so:
		// ind = ...___|_,_,_,_,_,_,_,3,8,6|_,_,_,_...
		//                el_mark = ^
		length_t el_mark = atomicSub(ell + src, 1) - 1;
		ind[off[src]+el_mark] = bud->getDst()[tid];
		seg[off[src]+el_mark] = src;
	}
}

template <typename T>
__global__ void initDeviceArray(T* mem, int32_t size, T value)
{
	int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		mem[idx] = value;
	}
}

// CPU Version - assume sorted index lists. 
int hostSingleIntersection (const vertexId_t ai, const length_t alen, const vertexId_t * a,
						    const vertexId_t bi, const length_t blen, const vertexId_t * b){
	length_t ka = 0, kb = 0,out = 0;
	if (!alen || !blen || a[alen-1] < b[0] || b[blen-1] < a[0])
    	return 0;

	while (1) {
    	if (ka >= alen || kb >= blen) break;
		vertexId_t va = a[ka],vb = b[kb];

	    // If you now that you don't have self edges then you don't need to check for them and you can get better performance.
		#if(0)
		    // Skip self-edges.
		    if ((va == ai)) {
		      ++ka;
		      if (ka >= alen) break;
		      va = a[ka];
		    }
		    if ((vb == bi)) {
		      ++kb;
		      if (kb >= blen) break;
		      vb = b[kb];
		    }
		#endif

	    if (va == vb) {
	     	++ka; ++kb; ++out;
	    }
	    else if (va < vb) {
	      ++ka;
	      while (ka < alen && a[ka] < vb) ++ka;
	    } else {
	      ++kb;
	      while (kb < blen && va > b[kb]) ++kb;
	    }
	}
	return out;
}

void hostCountTriangles (const vertexId_t nv, const length_t * off,
    const vertexId_t * ind, int * triNE, int64_t* allTriangles)
{
	int32_t edge=0;
	int64_t sum=0;
    for (vertexId_t src = 0; src < nv; src++)
    {
		length_t srcLen=off[src+1]-off[src];
		for(int iter=off[src]; iter<off[src+1]; iter++)
		{
			vertexId_t dest=ind[iter];
			length_t destLen=off[dest+1]-off[dest];			
			triNE[edge]= hostSingleIntersection (src, srcLen, ind+off[src],
													dest, destLen, ind+off[dest]);
			sum+=triNE[edge++];
		}
	}	
	*allTriangles=sum;
}

void hostCount2Triangles (const vertexId_t onv, const length_t * ooff,
    const vertexId_t * oind, const vertexId_t nv, const length_t * off,
    const vertexId_t * ind, int * triNE, int64_t* allTriangles)
{
	int32_t edge=0;
	int64_t sum=0;
    for (vertexId_t src = 0; src < nv; src++)
    {
		length_t srcLen=off[src+1]-off[src];
		for(int iter=off[src]; iter<off[src+1]; iter++)
		{
			vertexId_t dest=ind[iter];
			length_t destLen=ooff[dest+1]-ooff[dest];			
			triNE[edge]= hostSingleIntersection (src, srcLen, ind+off[src],
													dest, destLen, oind+ooff[dest]);
			sum+=triNE[edge++];
		}
	}	
	*allTriangles=sum;
}

void callDeviceNewTriangles(cuStinger& custing, BatchUpdate& bu, length_t nV, length_t* noff, vertexId_t* nind,
    triangle_t * const __restrict__ outPutTriangles, const int threads_per_block,
    const int number_blocks, const int shifter, const int thread_blocks, const int blockdim){

	// Calculate all new traingles regardless of repetition
	devicecuStingerNewTriangles<<<thread_blocks, blockdim>>>(custing.devicePtr(), bu.getDeviceBUD()->devicePtr(), outPutTriangles, threads_per_block,number_blocks,shifter);

	// Convert BUD to CSR
	// ==================
	length_t nv = custing.nv;

	// Allocate bud offset array and edgelist length
	length_t* d_boff = (length_t*) allocDeviceArray(nv+1, sizeof(length_t));
	length_t* d_ell = (length_t*) allocDeviceArray(nv+1, sizeof(length_t));

	dim3 numBlocks(1, 1);

	// Calculate edgelist lengths
	length_t batchsize = *(bu.getHostBUD()->getBatchSize());

	numBlocks.x = ceil((float)nv/(float)blockdim);
	initDeviceArray<<<numBlocks,blockdim>>>(d_ell, nv, 0);

	numBlocks.x = ceil((float)batchsize/(float)blockdim);
	calcEdgelistLengths<<<numBlocks,blockdim>>>(bu.getDeviceBUD()->devicePtr(), d_ell);

	vertexId_t* h_ell = (vertexId_t*) allocHostArray(nv+1, sizeof(vertexId_t));///
	copyArrayDeviceToHost(d_ell, h_ell, nv+1, sizeof(vertexId_t));///
	int64_t sumupd = sumTriangleArrayTEST(h_ell,nv);///

	// Calculate offsets by exclusive scan
	thrust::device_ptr<vertexId_t> dp_ell(d_ell);
	thrust::device_ptr<vertexId_t> dp_boff(d_boff);
	thrust::exclusive_scan(dp_ell, dp_ell+nv+1, dp_boff);

	// Make indices array and segment array
	length_t* scratchpad_l = (length_t*) allocHostArray(1, sizeof(length_t));/// not needed. same as batchsize
	copyArrayDeviceToHost(d_boff+nv, scratchpad_l, 1, sizeof(length_t));///
	length_t* h_boff = (length_t*) allocHostArray(nv+1, sizeof(length_t));///
	copyArrayDeviceToHost(d_boff, h_boff, nv+1, sizeof(length_t));///
	vertexId_t* d_bind = (vertexId_t*) allocDeviceArray(scratchpad_l[0], sizeof(vertexId_t));
	vertexId_t* d_bseg = (vertexId_t*) allocDeviceArray(scratchpad_l[0], sizeof(vertexId_t));

	// Populate indices array and segment array
	copyIndices<<<numBlocks,blockdim>>>(bu.getDeviceBUD()->devicePtr(), d_bind, d_bseg, d_boff, d_ell);

	// Sort the added edges
	thrust::device_ptr<vertexId_t> dp_bind(d_bind);
	thrust::device_ptr<vertexId_t> dp_bseg(d_bseg);
	thrust::stable_sort_by_key(dp_bind, dp_bind + scratchpad_l[0], dp_bseg);
	thrust::stable_sort_by_key(dp_bseg, dp_bseg + scratchpad_l[0], dp_bind);
	vertexId_t* h_bind = (vertexId_t*) allocHostArray(scratchpad_l[0], sizeof(vertexId_t));///
	vertexId_t* h_bseg = (vertexId_t*) allocHostArray(scratchpad_l[0], sizeof(vertexId_t));///
	copyArrayDeviceToHost(d_bind, h_bind, scratchpad_l[0], sizeof(vertexId_t));///
	copyArrayDeviceToHost(d_bseg, h_bseg, scratchpad_l[0], sizeof(vertexId_t));///
	// ==================
	// Done converting

	// Calculate triangles formed by only new edges
	triangle_t* d_3tri = (triangle_t*) allocDeviceArray(nv, sizeof(triangle_t));
	deviceBUThreeTriangles<<<thread_blocks,blockdim>>>(nv,d_boff, d_bind, d_3tri, threads_per_block,number_blocks,shifter);
	triangle_t* h_3tri = (triangle_t*) allocHostArray(batchsize, sizeof(triangle_t));///
	int64_t allTrianglesCPU=0;///
	// hostCountTriangles(nv, h_boff, h_bind, h_3tri, &allTrianglesCPU);///
	
	// Comparing boff and bind with CPU generated ones
		length_t* bell = (length_t*) allocHostArray(nv+1, sizeof(length_t));
		length_t* boff = (length_t*) allocHostArray(nv+1, sizeof(length_t));
		vertexId_t* bind = (vertexId_t*) allocHostArray(batchsize, sizeof(vertexId_t));
		vertexId_t* bseg = (vertexId_t*) allocHostArray(batchsize, sizeof(vertexId_t));
		vertexId_t* src = bu.getHostBUD()->getSrc();
		vertexId_t* dst = bu.getHostBUD()->getDst();
		triangle_t* threetri = (triangle_t*) allocHostArray(batchsize, sizeof(triangle_t));
		for (int i = 0; i < nv; ++i) bell[i] = 0;
		for (int i = 0; i < batchsize; ++i) {
			bell[src[i]]++;
		}
		boff[0] = 0;
		for (int i = 1; i < nv+1; ++i) {
			boff[i+1] = boff[i] + bell[i];
		}
		for (int i = 0; i < batchsize; ++i) {
			bell[src[i]]--;
			bind[ boff[src[i]] + bell[src[i]] ] = dst[i];
			bseg[ boff[src[i]] + bell[src[i]] ] = src[i];
		}
		thrust::stable_sort_by_key(bind, bind + batchsize, bseg);
		thrust::stable_sort_by_key(bseg, bseg + batchsize, bind);
		for (int i = 0; i < batchsize; ++i)
		{
			if (bind[i] != h_bind[i])
			{
				// printf("Mismatch here\n");
			}
		}
		for (int i = 0; i < 10; ++i)
		{
			printf("seg compare %d %d\n", bseg[i], h_bseg[i]);
		}
		hostCountTriangles(nv, boff, bind, threetri, &allTrianglesCPU);
		printf("cpu 3tri %d\n", allTrianglesCPU);///
		copyArrayHostToDevice(bind, d_bind, batchsize, sizeof(vertexId_t));
		int64_t all2TrianglesCPU=0;///
		hostCount2Triangles(nv, noff, nind, nv, boff, bind, threetri, &all2TrianglesCPU);
		printf("cpu 2tri %d\n", all2TrianglesCPU);///

	// Calculate triangles formed by two new and one old edges
	triangle_t* d_2tri = (triangle_t*) allocDeviceArray(nv, sizeof(triangle_t));
	deviceBUTwoCUOneTriangles<<<thread_blocks,blockdim>>>(custing.devicePtr(),nv,d_boff, d_bind, d_2tri, threads_per_block,number_blocks,shifter);

	// TESTING. Remove after done
	// triangle_t* h_3tri = (triangle_t*) allocHostArray(nv, sizeof(triangle_t));
	triangle_t* h_2tri = (triangle_t*) allocHostArray(nv, sizeof(triangle_t));
	hipMemcpy(h_3tri, d_3tri, sizeof(triangle_t)*nv, hipMemcpyDeviceToHost);
	hipMemcpy(h_2tri, d_2tri, sizeof(triangle_t)*nv, hipMemcpyDeviceToHost);
	int64_t sum3 = sumTriangleArrayTEST(h_3tri,nv);
	int64_t sum2 = sumTriangleArrayTEST(h_2tri,nv);
	printf("Sum2=%d \n Sum3=%d\n",sum2,sum3);
}

