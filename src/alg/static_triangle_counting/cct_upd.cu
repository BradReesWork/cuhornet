#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <inttypes.h>

#include "cct.hpp"
 
__device__ void initialize(const vertexId_t diag_id, const length_t u_len, length_t v_len,
    length_t* const __restrict__ u_min, length_t* const __restrict__ u_max,
    length_t* const __restrict__ v_min, length_t* const __restrict__ v_max,
    int* const __restrict__ found)
{
	if (diag_id == 0){
		*u_min=*u_max=*v_min=*v_max=0;
		*found=1;
	}
	else if (diag_id < u_len){
		*u_min=0; *u_max=diag_id;
		*v_max=diag_id;*v_min=0;
	}
	else if (diag_id < v_len){
		*u_min=0; *u_max=u_len;
		*v_max=diag_id;*v_min=diag_id-u_len;
	}
	else{
		*u_min=diag_id-v_len; *u_max=u_len;
		*v_min=diag_id-u_len; *v_max=v_len;
	}
}

__device__ void workPerThread(const length_t uLength, const length_t vLength, 
	const int threadsPerIntersection, const int threadId,
    int * const __restrict__ outWorkPerThread, int * const __restrict__ outDiagonalId){
  int totalWork = uLength + vLength;
  int remainderWork = totalWork%threadsPerIntersection;
  int workPerThread = totalWork/threadsPerIntersection;

  int longDiagonals  = (threadId > remainderWork) ? remainderWork:threadId;
  int shortDiagonals = (threadId > remainderWork) ? (threadId - remainderWork):0;

  *outDiagonalId = ((workPerThread+1)*longDiagonals) + (workPerThread*shortDiagonals);
  *outWorkPerThread = workPerThread + (threadId < remainderWork);
}

__device__ void bSearch(unsigned int found, const vertexId_t diagonalId,
    vertexId_t const * const __restrict__ uNodes, vertexId_t const * const __restrict__ vNodes,
    length_t const * const __restrict__ uLength, 
    length_t * const __restrict__ outUMin, length_t * const __restrict__ outUMax,
    length_t * const __restrict__ outVMin, length_t * const __restrict__ outVMax,    
    length_t * const __restrict__ outUCurr,
    length_t * const __restrict__ outVCurr){
  	length_t length;
	
	while(!found) {
	    *outUCurr = (*outUMin + *outUMax)>>1;
	    *outVCurr = diagonalId - *outUCurr;
	    if(*outVCurr >= *outVMax){
			length = *outUMax - *outUMin;
			if(length == 1){
				found = 1;
				continue;
			}
	    }

	    unsigned int comp1 = uNodes[*outUCurr] > vNodes[*outVCurr-1];
	    unsigned int comp2 = uNodes[*outUCurr-1] > vNodes[*outVCurr];
	    if(comp1 && !comp2){
			found = 1;
	    }
	    else if(comp1){
	      *outVMin = *outVCurr;
	      *outUMax = *outUCurr;
	    }
	    else{
	      *outVMax = *outVCurr;
	      *outUMin = *outUCurr;
	    }
  	}

	if((*outVCurr >= *outVMax) && (length == 1) && (*outVCurr > 0) &&
	(*outUCurr > 0) && (*outUCurr < (*uLength - 1))){
		unsigned int comp1 = uNodes[*outUCurr] > vNodes[*outVCurr - 1];
		unsigned int comp2 = uNodes[*outUCurr - 1] > vNodes[*outVCurr];
		if(!comp1 && !comp2){(*outUCurr)++; (*outVCurr)--;}
	}
}

__device__ int fixStartPoint(const length_t uLength, const length_t vLength,
    length_t * const __restrict__ uCurr, length_t * const __restrict__ vCurr,
    vertexId_t const * const __restrict__ uNodes, vertexId_t const * const __restrict__ vNodes){
	
	unsigned int uBigger = (*uCurr > 0) && (*vCurr < vLength) && (uNodes[*uCurr-1] == vNodes[*vCurr]);
	unsigned int vBigger = (*vCurr > 0) && (*uCurr < uLength) && (vNodes[*vCurr-1] == uNodes[*uCurr]);
	*uCurr += vBigger;
	*vCurr += uBigger;
	return (uBigger + vBigger);
}

__device__ void intersectCount(const length_t uLength, const length_t vLength,
    vertexId_t const * const __restrict__ uNodes, vertexId_t const * const __restrict__ vNodes,
    length_t * const __restrict__ uCurr, length_t * const __restrict__ vCurr,
    int * const __restrict__ workIndex, int * const __restrict__ workPerThread,
    int * const __restrict__ triangles, int found)
{
  if((*uCurr < uLength) && (*vCurr < vLength)){
    int comp;
    while(*workIndex < *workPerThread){
		comp = uNodes[*uCurr] - vNodes[*vCurr];
		*triangles += (comp == 0);
		*uCurr += (comp <= 0);
		*vCurr += (comp >= 0);
		*workIndex += (comp == 0) + 1;

		if((*vCurr == vLength) || (*uCurr == uLength)){
			break;
		}
    }
    *triangles -= ((comp == 0) && (*workIndex > *workPerThread) && (found));
  }
}


// u_len < v_len
__device__ triangle_t count_triangles(vertexId_t u, vertexId_t const * const __restrict__ u_nodes, length_t u_len,
    vertexId_t v, vertexId_t const * const __restrict__ v_nodes, length_t v_len, int threads_per_block,
    volatile vertexId_t* __restrict__ firstFound, int tId)
{
	// Partitioning the work to the multiple thread of a single GPU processor. The threads should get a near equal number of the elements to Tersect - this number will be off by 1.
	int work_per_thread, diag_id;
	workPerThread(u_len, v_len, threads_per_block, tId, &work_per_thread, &diag_id);
	triangle_t triangles = 0;
	int work_index = 0,found=0;
	length_t u_min,u_max,v_min,v_max,u_curr,v_curr;

	firstFound[tId]=0;

	if(work_per_thread>0){
		// For the binary search, we are figuring out the initial poT of search.
		initialize(diag_id, u_len, v_len,&u_min, &u_max,&v_min, &v_max,&found);
    	u_curr = 0; v_curr = 0;

	    bSearch(found, diag_id, u_nodes, v_nodes, &u_len, &u_min, &u_max, &v_min,
        &v_max, &u_curr, &v_curr);

    	int sum = fixStartPoint(u_len, v_len, &u_curr, &v_curr, u_nodes, v_nodes);
    	work_index += sum;
	    if(tId > 0)
	      firstFound[tId-1] = sum;
	    triangles += sum;
	    intersectCount(u_len, v_len, u_nodes, v_nodes, &u_curr, &v_curr,
	        &work_index, &work_per_thread, &triangles, firstFound[tId]);
	}
	return triangles;
}

__device__ void workPerBlock(const length_t numVertices,
    length_t * const __restrict__ outMpStart,
    length_t * const __restrict__ outMpEnd, int blockSize)
{
	length_t verticesPerMp = numVertices/gridDim.x;
	length_t remainderBlocks = numVertices % gridDim.x;
	length_t extraVertexBlocks = (blockIdx.x > remainderBlocks)? remainderBlocks:blockIdx.x;
	length_t regularVertexBlocks = (blockIdx.x > remainderBlocks)? blockIdx.x - remainderBlocks:0;

	length_t mpStart = ((verticesPerMp+1)*extraVertexBlocks) + (verticesPerMp*regularVertexBlocks);
	*outMpStart = mpStart;
	*outMpEnd = mpStart + verticesPerMp + (blockIdx.x < remainderBlocks);
}


__global__ void devicecuStingerNewTriangles(cuStinger* custing, BatchUpdateData *bud,
    triangle_t * const __restrict__ outPutTriangles, const int threads_per_block,
    const int number_blocks, const int shifter)
{
	length_t batchSize = *(bud->getBatchSize());
	// Partitioning the work to the multiple thread of a single GPU processor. The threads should get a near equal number of the elements to intersect - this number will be off by no more than one.
	int tx = threadIdx.x;
 	length_t this_mp_start, this_mp_stop;

	const int blockSize = blockDim.x;
	workPerBlock(batchSize, &this_mp_start, &this_mp_stop, blockSize);

	__shared__ vertexId_t firstFound[1024];

	length_t adj_offset=tx>>shifter;
	length_t* firstFoundPos=firstFound + (adj_offset<<shifter);
	for (length_t edge = this_mp_start+adj_offset; edge < this_mp_stop; edge+=number_blocks){
		vertexId_t src = bud->getSrc()[edge];
		vertexId_t dest= bud->getDst()[edge];
		length_t srcLen=custing->dVD->getUsed()[src];
		length_t destLen=custing->dVD->getUsed()[dest];

		bool avoidCalc = (src == dest) || (destLen < 2) || (srcLen < 2);
		if(avoidCalc)
			continue;

		bool sourceSmaller = (srcLen<destLen);
        vertexId_t small = sourceSmaller? src : dest;
        vertexId_t large = sourceSmaller? dest : src;
        length_t small_len = sourceSmaller? srcLen : destLen;
        length_t large_len = sourceSmaller? destLen : srcLen;

        const vertexId_t* small_ptr = custing->dVD->getAdj()[small]->dst;
        const vertexId_t* large_ptr = custing->dVD->getAdj()[large]->dst;

		triangle_t tCount = count_triangles(small, small_ptr, small_len,
								large,large_ptr, large_len,
								threads_per_block,firstFoundPos,
								tx%threads_per_block);

		atomicAdd(outPutTriangles + src, tCount);
	}
}

void callDeviceNewTriangles(cuStinger& custing, BatchUpdate& bu,
    triangle_t * const __restrict__ outPutTriangles, const int threads_per_block,
    const int number_blocks, const int shifter, const int thread_blocks, const int blockdim){

	devicecuStingerNewTriangles<<<thread_blocks, blockdim>>>(custing.devicePtr(), bu.getDeviceBUD()->devicePtr(), outPutTriangles, threads_per_block,number_blocks,shifter);
}



