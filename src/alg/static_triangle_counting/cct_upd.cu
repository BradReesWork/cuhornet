#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <inttypes.h>

#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/remove.h>

#include "cct.hpp"
#include "utils.hpp"

#include "kernel_segsort.hxx"

using namespace mgpu;

__device__ void conditionalWarpReduce(volatile triangle_t* sharedData,int blockSize,int dataLength){
  if(blockSize >= dataLength){
    if(threadIdx.x < (dataLength/2))
    {sharedData[threadIdx.x] += sharedData[threadIdx.x+(dataLength/2)];}
    __syncthreads();
  }
}

__device__ void warpReduce(triangle_t* __restrict__ outDataPtr,
    volatile triangle_t* __restrict__ sharedData,int blockSize){
  conditionalWarpReduce(sharedData,blockSize,64);
  conditionalWarpReduce(sharedData,blockSize,32);
  conditionalWarpReduce(sharedData,blockSize,16);
  conditionalWarpReduce(sharedData,blockSize,8);
  conditionalWarpReduce(sharedData,blockSize,4);
  if(threadIdx.x == 0)
    {*outDataPtr= sharedData[0] + sharedData[1];}
  __syncthreads();
}

__device__ void conditionalReduce(volatile triangle_t* __restrict__ sharedData,int blockSize,int dataLength){
	if(blockSize >= dataLength){
		if(threadIdx.x < (dataLength/2))
		{sharedData[threadIdx.x] += sharedData[threadIdx.x+(dataLength/2)];}
		__syncthreads();
	}
	if((blockSize < dataLength) && (blockSize > (dataLength/2))){
		if(threadIdx.x+(dataLength/2) < blockSize){
			sharedData[threadIdx.x] += sharedData[threadIdx.x+(dataLength/2)];
		}
		__syncthreads();
	}
}

__device__ void blockReduce(triangle_t* __restrict__ outGlobalDataPtr,
    volatile triangle_t* __restrict__ sharedData,int blockSize){
  __syncthreads();
  conditionalReduce(sharedData,blockSize,1024);
  conditionalReduce(sharedData,blockSize,512);
  conditionalReduce(sharedData,blockSize,256);
  conditionalReduce(sharedData,blockSize,128);

  warpReduce(outGlobalDataPtr, sharedData, blockSize);
  __syncthreads();
}

__device__ void initialize(const vertexId_t diag_id, const length_t u_len, length_t v_len,
    length_t* const __restrict__ u_min, length_t* const __restrict__ u_max,
    length_t* const __restrict__ v_min, length_t* const __restrict__ v_max,
    int* const __restrict__ found)
{
	if (diag_id == 0){
		*u_min=*u_max=*v_min=*v_max=0;
		*found=1;
	}
	else if (diag_id < u_len){
		*u_min=0; *u_max=diag_id;
		*v_max=diag_id;*v_min=0;
	}
	else if (diag_id < v_len){
		*u_min=0; *u_max=u_len;
		*v_max=diag_id;*v_min=diag_id-u_len;
	}
	else{
		*u_min=diag_id-v_len; *u_max=u_len;
		*v_min=diag_id-u_len; *v_max=v_len;
	}
}

__device__ void workPerThread(const length_t uLength, const length_t vLength, 
	const int threadsPerIntersection, const int threadId,
    int * const __restrict__ outWorkPerThread, int * const __restrict__ outDiagonalId){
  int totalWork = uLength + vLength;
  int remainderWork = totalWork%threadsPerIntersection;
  int workPerThread = totalWork/threadsPerIntersection;

  int longDiagonals  = (threadId > remainderWork) ? remainderWork:threadId;
  int shortDiagonals = (threadId > remainderWork) ? (threadId - remainderWork):0;

  *outDiagonalId = ((workPerThread+1)*longDiagonals) + (workPerThread*shortDiagonals);
  *outWorkPerThread = workPerThread + (threadId < remainderWork);
}

__device__ void bSearch(unsigned int found, const vertexId_t diagonalId,
    vertexId_t const * const __restrict__ uNodes, vertexId_t const * const __restrict__ vNodes,
    length_t const * const __restrict__ uLength, 
    length_t * const __restrict__ outUMin, length_t * const __restrict__ outUMax,
    length_t * const __restrict__ outVMin, length_t * const __restrict__ outVMax,    
    length_t * const __restrict__ outUCurr,
    length_t * const __restrict__ outVCurr){
  	length_t length;
	
	while(!found) {
	    *outUCurr = (*outUMin + *outUMax)>>1;
	    *outVCurr = diagonalId - *outUCurr;
	    if(*outVCurr >= *outVMax){
			length = *outUMax - *outUMin;
			if(length == 1){
				found = 1;
				continue;
			}
	    }

	    unsigned int comp1 = uNodes[*outUCurr] > vNodes[*outVCurr-1];
	    unsigned int comp2 = uNodes[*outUCurr-1] > vNodes[*outVCurr];
	    if(comp1 && !comp2){
			found = 1;
	    }
	    else if(comp1){
	      *outVMin = *outVCurr;
	      *outUMax = *outUCurr;
	    }
	    else{
	      *outVMax = *outVCurr;
	      *outUMin = *outUCurr;
	    }
  	}

	if((*outVCurr >= *outVMax) && (length == 1) && (*outVCurr > 0) &&
	(*outUCurr > 0) && (*outUCurr < (*uLength - 1))){
		unsigned int comp1 = uNodes[*outUCurr] > vNodes[*outVCurr - 1];
		unsigned int comp2 = uNodes[*outUCurr - 1] > vNodes[*outVCurr];
		if(!comp1 && !comp2){(*outUCurr)++; (*outVCurr)--;}
	}
}

__device__ int fixStartPoint(const length_t uLength, const length_t vLength,
    length_t * const __restrict__ uCurr, length_t * const __restrict__ vCurr,
    vertexId_t const * const __restrict__ uNodes, vertexId_t const * const __restrict__ vNodes){
	
	unsigned int uBigger = (*uCurr > 0) && (*vCurr < vLength) && (uNodes[*uCurr-1] == vNodes[*vCurr]);
	unsigned int vBigger = (*vCurr > 0) && (*uCurr < uLength) && (vNodes[*vCurr-1] == uNodes[*uCurr]);
	*uCurr += vBigger;
	*vCurr += uBigger;
	return (uBigger + vBigger);
}

__device__ void intersectCount(const length_t uLength, const length_t vLength,
    vertexId_t const * const __restrict__ uNodes, vertexId_t const * const __restrict__ vNodes,
    length_t * const __restrict__ uCurr, length_t * const __restrict__ vCurr,
    int * const __restrict__ workIndex, int * const __restrict__ workPerThread,
    int * const __restrict__ triangles, int found,
    vertexId_t const * const __restrict__ uMask, vertexId_t const * const __restrict__ vMask,
    const bool uMasked, const bool vMasked)
{
  if((*uCurr < uLength) && (*vCurr < vLength)){
    int comp;
    int vmask;
    int umask;
    while(*workIndex < *workPerThread){
    	vmask = (vMasked) ? vMask[*vCurr] : 0;
        umask = (uMasked) ? uMask[*uCurr] : 0;
		comp = uNodes[*uCurr] - vNodes[*vCurr];
		*triangles += (comp == 0 && !umask && !vmask);
		*uCurr += (comp <= 0 && !vmask) || umask;
		*vCurr += (comp >= 0 && !umask) || vmask;
		*workIndex += (comp == 0&& !umask && !vmask) + 1;

		if((*vCurr == vLength) || (*uCurr == uLength)){
			break;
		}
    }
    *triangles -= ((comp == 0) && (*workIndex > *workPerThread) && (found));
  }
}


// u_len < v_len
__device__ triangle_t count_triangles(vertexId_t u, vertexId_t const * const __restrict__ u_nodes, length_t u_len,
    vertexId_t v, vertexId_t const * const __restrict__ v_nodes, length_t v_len, int threads_per_block,
    volatile vertexId_t* __restrict__ firstFound, int tId,
    vertexId_t const * const __restrict__ uMask, vertexId_t const * const __restrict__ vMask,
    const bool uMasked, const bool vMasked)
{
	// Partitioning the work to the multiple thread of a single GPU processor. The threads should get a near equal number of the elements to Tersect - this number will be off by 1.
	int work_per_thread, diag_id;
	workPerThread(u_len, v_len, threads_per_block, tId, &work_per_thread, &diag_id);
	triangle_t triangles = 0;
	int work_index = 0,found=0;
	length_t u_min,u_max,v_min,v_max,u_curr,v_curr;

	firstFound[tId]=0;

	if(work_per_thread>0){
		// For the binary search, we are figuring out the initial poT of search.
		initialize(diag_id, u_len, v_len,&u_min, &u_max,&v_min, &v_max,&found);
    	u_curr = 0; v_curr = 0;

	    bSearch(found, diag_id, u_nodes, v_nodes, &u_len, &u_min, &u_max, &v_min,
        &v_max, &u_curr, &v_curr);

    	int sum = fixStartPoint(u_len, v_len, &u_curr, &v_curr, u_nodes, v_nodes);
    	work_index += sum;
	    if(tId > 0)
	      firstFound[tId-1] = sum;
	    triangles += sum;
	    intersectCount(u_len, v_len, u_nodes, v_nodes, &u_curr, &v_curr,
	        &work_index, &work_per_thread, &triangles, firstFound[tId], 
	        uMask, vMask, uMasked, vMasked);
	}
	return triangles;
}

template <bool uMasked, bool vMasked, bool subtract>
__device__ void intersectCount_nc(const length_t uLength, const length_t vLength,
    vertexId_t const * const __restrict__ uNodes, vertexId_t const * const __restrict__ vNodes,
    length_t * const __restrict__ uCurr, length_t * const __restrict__ vCurr,
    int * const __restrict__ workIndex, int * const __restrict__ workPerThread,
    int * const __restrict__ triangles, int found, triangle_t * const __restrict__ outPutTriangles, 
    vertexId_t const * const __restrict__ uMask, vertexId_t const * const __restrict__ vMask)
    // const bool uMasked, const bool vMasked, const bool subtract)
{
  if((*uCurr < uLength) && (*vCurr < vLength)){
    int comp;
    int vmask;
    int umask;
    while(*workIndex < *workPerThread){
    	vmask = (vMasked) ? vMask[*vCurr] : 0;
        umask = (uMasked) ? uMask[*uCurr] : 0;
		comp = uNodes[*uCurr] - vNodes[*vCurr];
		*triangles += (comp == 0 && !umask && !vmask);
		if (comp == 0 && !umask && !vmask)
			if (subtract) atomicSub(outPutTriangles + uNodes[*uCurr], 1);
			else atomicAdd(outPutTriangles + uNodes[*uCurr], 1);
		*uCurr += (comp <= 0 && !vmask) || umask;
		*vCurr += (comp >= 0 && !umask) || vmask;
		*workIndex += (comp == 0&& !umask && !vmask) + 1;

		if((*vCurr == vLength) || (*uCurr == uLength)){
			break;
		}
    }
    *triangles -= ((comp == 0) && (*workIndex > *workPerThread) && (found));
  }
}


// u_len < v_len
template <bool uMasked, bool vMasked, bool subtract>
__device__ triangle_t count_triangles_nc(vertexId_t u, vertexId_t const * const __restrict__ u_nodes, length_t u_len,
    vertexId_t v, vertexId_t const * const __restrict__ v_nodes, length_t v_len, int threads_per_block,
    volatile vertexId_t* __restrict__ firstFound, int tId, triangle_t * const __restrict__ outPutTriangles,
    vertexId_t const * const __restrict__ uMask, vertexId_t const * const __restrict__ vMask)
    // const bool uMasked, const bool vMasked, const bool subtract)
{
	// Partitioning the work to the multiple thread of a single GPU processor. The threads should get a near equal number of the elements to Tersect - this number will be off by 1.
	int work_per_thread, diag_id;
	workPerThread(u_len, v_len, threads_per_block, tId, &work_per_thread, &diag_id);
	triangle_t triangles = 0;
	int work_index = 0,found=0;
	length_t u_min,u_max,v_min,v_max,u_curr,v_curr;

	firstFound[tId]=0;

	if(work_per_thread>0){
		// For the binary search, we are figuring out the initial poT of search.
		initialize(diag_id, u_len, v_len,&u_min, &u_max,&v_min, &v_max,&found);
    	u_curr = 0; v_curr = 0;

	    bSearch(found, diag_id, u_nodes, v_nodes, &u_len, &u_min, &u_max, &v_min,
        &v_max, &u_curr, &v_curr);

    	int sum = fixStartPoint(u_len, v_len, &u_curr, &v_curr, u_nodes, v_nodes);
    	work_index += sum;
	    if(tId > 0)
	      firstFound[tId-1] = sum;
	    triangles += sum;
	    intersectCount_nc<uMasked, vMasked, subtract>(u_len, v_len, u_nodes, v_nodes, &u_curr, &v_curr,
	        &work_index, &work_per_thread, &triangles, firstFound[tId], outPutTriangles, 
	        uMask, vMask);
	}
	return triangles;
}

__device__ void workPerBlock(const length_t numVertices,
    length_t * const __restrict__ outMpStart,
    length_t * const __restrict__ outMpEnd, int blockSize)
{
	length_t verticesPerMp = numVertices/gridDim.x;
	length_t remainderBlocks = numVertices % gridDim.x;
	length_t extraVertexBlocks = (blockIdx.x > remainderBlocks)? remainderBlocks:blockIdx.x;
	length_t regularVertexBlocks = (blockIdx.x > remainderBlocks)? blockIdx.x - remainderBlocks:0;

	length_t mpStart = ((verticesPerMp+1)*extraVertexBlocks) + (verticesPerMp*regularVertexBlocks);
	*outMpStart = mpStart;
	*outMpEnd = mpStart + verticesPerMp + (blockIdx.x < remainderBlocks);
}


__global__ void devicecuStingerNewTriangles(cuStinger* custing, BatchUpdateData *bud,
    triangle_t * const __restrict__ outPutTriangles, const int threads_per_block,
    const int number_blocks, const int shifter)
{
	length_t batchSize = *(bud->getBatchSize());
	// Partitioning the work to the multiple thread of a single GPU processor. The threads should get a near equal number of the elements to intersect - this number will be off by no more than one.
	int tx = threadIdx.x;
 	length_t this_mp_start, this_mp_stop;

	length_t *d_off = bud->getOffsets();
	vertexId_t * d_ind = bud->getDst();
	vertexId_t * d_seg = bud->getSrc();

	const int blockSize = blockDim.x;
	workPerBlock(batchSize, &this_mp_start, &this_mp_stop, blockSize);

	__shared__ vertexId_t firstFound[1024];

	length_t adj_offset=tx>>shifter;
	length_t* firstFoundPos=firstFound + (adj_offset<<shifter);
	for (length_t edge = this_mp_start+adj_offset; edge < this_mp_stop; edge+=number_blocks){
		if (bud->getIndDuplicate()[edge]==1) // this means it's a duplicate edge
			continue;

		vertexId_t src = d_seg[edge];
		vertexId_t dest= d_ind[edge];

		length_t srcLen=custing->dVD->getUsed()[src];
		length_t destLen=custing->dVD->getUsed()[dest];

		bool avoidCalc = (src == dest) || (destLen < 2) || (srcLen < 2);
		if(avoidCalc)
			continue;

		bool sourceSmaller = (srcLen<destLen);
        vertexId_t small = sourceSmaller? src : dest;
        vertexId_t large = sourceSmaller? dest : src;
        length_t small_len = sourceSmaller? srcLen : destLen;
        length_t large_len = sourceSmaller? destLen : srcLen;

        const vertexId_t* small_ptr = custing->dVD->getAdj()[small]->dst;
        const vertexId_t* large_ptr = custing->dVD->getAdj()[large]->dst;

		triangle_t tCount = count_triangles_nc<false, false, false>(
								small, small_ptr, small_len,
								large,large_ptr, large_len,
								threads_per_block,firstFoundPos,
								tx%threads_per_block, outPutTriangles,
								NULL, NULL);

		atomicAdd(outPutTriangles + src, tCount);
		atomicAdd(outPutTriangles + dest, tCount);
		__syncthreads();
	}
}

template <typename T>
T sumTriangleArrayTEST(T* h_triangles, vertexId_t nv){	
	T sum=0;
	for(vertexId_t sd=0; sd<(nv);sd++){
	  sum+=h_triangles[sd];
	}
	return sum;
}

__global__ void deviceBUThreeTriangles (BatchUpdateData *bud,
    triangle_t * const __restrict__ outPutTriangles, const int threads_per_block,
    const int number_blocks, const int shifter)
{
	length_t batchsize = *(bud->getBatchSize());
	// Partitioning the work to the multiple thread of a single GPU processor. The threads should get a near equal number of the elements to intersect - this number will be off by no more than one.
	int tx = threadIdx.x;
 	length_t this_mp_start, this_mp_stop;

	length_t *d_off = bud->getOffsets();
	vertexId_t * d_ind = bud->getDst();
	vertexId_t * d_seg = bud->getSrc();

	const int blockSize = blockDim.x;
	workPerBlock(batchsize, &this_mp_start, &this_mp_stop, blockSize);

	__shared__ vertexId_t firstFound[1024];

	length_t adj_offset=tx>>shifter;
	length_t* firstFoundPos=firstFound + (adj_offset<<shifter);
	for (length_t edge = this_mp_start+adj_offset; edge < this_mp_stop; edge+=number_blocks){
		if (bud->getIndDuplicate()[edge]) // this means it's a duplicate edge
			continue;
			
		vertexId_t src = d_seg[edge];
		vertexId_t dest= d_ind[edge];
		length_t srcLen= d_off[src+1] - d_off[src];
		length_t destLen=d_off[dest+1] - d_off[dest];

		bool avoidCalc = (src == dest) || (destLen < 2) || (srcLen < 2);
		if(avoidCalc)
			continue;

		bool sourceSmaller = (srcLen<destLen);
        vertexId_t small = sourceSmaller? src : dest;
        vertexId_t large = sourceSmaller? dest : src;
        length_t small_len = sourceSmaller? srcLen : destLen;
        length_t large_len = sourceSmaller? destLen : srcLen;

        vertexId_t const * const small_ptr = d_ind + d_off[small];
        vertexId_t const * const large_ptr = d_ind + d_off[large];
        vertexId_t const * const small_mask_ptr = bud->getIndDuplicate() + d_off[small];
        vertexId_t const * const large_mask_ptr = bud->getIndDuplicate() + d_off[large];

		triangle_t tCount = count_triangles(small, small_ptr, small_len,
								large,large_ptr, large_len,
								threads_per_block,firstFoundPos,
								tx%threads_per_block,
								small_mask_ptr, large_mask_ptr, true, true);

		atomicAdd(outPutTriangles + src, tCount);
		__syncthreads();
	}
}

__global__ void deviceBUTwoCUOneTriangles (BatchUpdateData *bud, cuStinger* custing,
    triangle_t * const __restrict__ outPutTriangles, const int threads_per_block,
    const int number_blocks, const int shifter)
{
	length_t batchsize = *(bud->getBatchSize());
	// Partitioning the work to the multiple thread of a single GPU processor. The threads should get a near equal number of the elements to intersect - this number will be off by no more than one.
	int tx = threadIdx.x;
 	vertexId_t this_mp_start, this_mp_stop;

	length_t *d_off = bud->getOffsets();
	vertexId_t * d_ind = bud->getDst();
	vertexId_t * d_seg = bud->getSrc();

	const int blockSize = blockDim.x;
	workPerBlock(batchsize, &this_mp_start, &this_mp_stop, blockSize);

	__shared__ vertexId_t firstFound[1024];

	length_t adj_offset=tx>>shifter;
	length_t* firstFoundPos=firstFound + (adj_offset<<shifter);
	for (length_t edge = this_mp_start+adj_offset; edge < this_mp_stop; edge+=number_blocks){
		if (bud->getIndDuplicate()[edge]) // this means it's a duplicate edge
			continue;
			
		vertexId_t src = bud->getSrc()[edge];
		vertexId_t dest= bud->getDst()[edge];
		length_t srcLen= d_off[src+1] - d_off[src];
		length_t destLen=custing->dVD->getUsed()[dest];

		bool avoidCalc = (src == dest) || (destLen < 2) || (srcLen < 2);
		if(avoidCalc)
			continue;

        vertexId_t const * const src_ptr = d_ind + d_off[src];
        vertexId_t const * const src_mask_ptr = bud->getIndDuplicate() + d_off[src];
        vertexId_t const * const dst_ptr = custing->dVD->getAdj()[dest]->dst;

		bool sourceSmaller = (srcLen<destLen);
        vertexId_t small = sourceSmaller? src : dest;
        vertexId_t large = sourceSmaller? dest : src;
        length_t small_len = sourceSmaller? srcLen : destLen;
        length_t large_len = sourceSmaller? destLen : srcLen;

        vertexId_t const * const small_ptr = sourceSmaller? src_ptr : dst_ptr;
        vertexId_t const * const small_mask_ptr = sourceSmaller? src_mask_ptr : NULL;
        vertexId_t const * const large_ptr = sourceSmaller? dst_ptr : src_ptr;
        vertexId_t const * const large_mask_ptr = sourceSmaller? NULL : src_mask_ptr;

		triangle_t tCount = (sourceSmaller)?
								count_triangles_nc<true, false, true>(
								small, small_ptr, small_len,
								large,large_ptr, large_len,
								threads_per_block,firstFoundPos,
								tx%threads_per_block, outPutTriangles,
								small_mask_ptr, large_mask_ptr):
								count_triangles_nc<false, true, true>(
								small, small_ptr, small_len,
								large,large_ptr, large_len,
								threads_per_block,firstFoundPos,
								tx%threads_per_block, outPutTriangles,
								small_mask_ptr, large_mask_ptr )
							;

		atomicSub(outPutTriangles + src, tCount);
		atomicSub(outPutTriangles + dest, tCount);
		__syncthreads();
	}
}


__global__ void calcEdgelistLengths(BatchUpdateData *bud, length_t* const __restrict__ ell){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	length_t batchSize = *(bud->getBatchSize());
	if (tid < batchSize) {
		vertexId_t src = bud->getSrc()[tid];
		atomicAdd(ell+src, 1);
	}
}

__global__ void copyIndices(BatchUpdateData *bud, vertexId_t* const __restrict__ ind,
	vertexId_t* const __restrict__ seg,	length_t* const __restrict__ off,
	length_t* const __restrict__ ell){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	length_t batchSize = *(bud->getBatchSize());
	if (tid < batchSize)
	{
		vertexId_t src = bud->getSrc()[tid];
		// Start filling up from the end of the edge list like so:
		// ind = ...___|_,_,_,_,_,_,_,3,8,6|_,_,_,_...
		//                el_mark = ^
		length_t el_mark = atomicSub(ell + src, 1) - 1;
		ind[off[src]+el_mark] = bud->getDst()[tid];
		seg[off[src]+el_mark] = src;
	}
}

template <typename T>
__global__ void initDeviceArray(T* mem, int32_t size, T value)
{
	int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		mem[idx] = value;
	}
}

__device__ void isort(vertexId_t* const __restrict__ u, length_t ell) {
	vertexId_t *v;
	vertexId_t w;
	for (int i = 0; i < ell; ++i) {
		v = u+i;
		while (v != u && *v < *(v-1)) {
			w = *v;
			*v = *(v-1);
			*(v-1) = w;
			v--;
		}
	}
}

__global__ void iSortAll(vertexId_t* const __restrict__ ind,
	length_t* const __restrict__ off, length_t nv) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < nv) {
		isort( &ind[ off[tid] ], off[tid+1] - off[tid]);
	}
}

void callDeviceNewTriangles(cuStinger& custing, BatchUpdate& bu, 
    triangle_t * const __restrict__ outPutTriangles, const int threads_per_block,
    const int number_blocks, const int shifter, const int thread_blocks, const int blockdim,
    triangle_t * const __restrict__ h_triangles, triangle_t * const __restrict__ h_triangles_t)
{
	hipEvent_t ce_start,ce_stop;

	dim3 numBlocks(1, 1);

	length_t batchsize = *(bu.getHostBUD()->getBatchSize());
	length_t nv = *(bu.getHostBUD()->getNumVertices());

	numBlocks.x = ceil((float)(batchsize*threads_per_block)/(float)blockdim);

	// Calculate all new traingles regardless of repetition
		start_clock(ce_start, ce_stop);
		devicecuStingerNewTriangles<<<numBlocks, blockdim>>>(custing.devicePtr(), bu.getDeviceBUD()->devicePtr(), outPutTriangles, threads_per_block,number_blocks,shifter);
		printf("\n%s <%d> %f\n", __FUNCTION__, __LINE__, end_clock(ce_start, ce_stop));

	// Calculate triangles formed by only new edges
		start_clock(ce_start, ce_stop);
		deviceBUThreeTriangles<<<numBlocks,blockdim>>>(bu.getDeviceBUD()->devicePtr(), outPutTriangles, threads_per_block,number_blocks,shifter);
		printf("\n%s <%d> %f\n", __FUNCTION__, __LINE__, end_clock(ce_start, ce_stop));
	
	// Calculate triangles formed by two new edges
		start_clock(ce_start, ce_stop);
		deviceBUTwoCUOneTriangles<<<numBlocks,blockdim>>>(bu.getDeviceBUD()->devicePtr(),custing.devicePtr(), outPutTriangles, threads_per_block,number_blocks,shifter);
		printf("\n%s <%d> %f\n", __FUNCTION__, __LINE__, end_clock(ce_start, ce_stop));
}

void testSort(length_t nv, BatchUpdate& bu,	const int blockdim){

	hipEvent_t ce_start,ce_stop;
	length_t batchsize = *(bu.getHostBUD()->getBatchSize());

	dim3 numBlocks(1, 1);

	// iSort approach =============================================
	start_clock(ce_start, ce_stop);
	vertexId_t* d_bind = (vertexId_t*) allocDeviceArray(batchsize, sizeof(vertexId_t));
	vertexId_t* d_bseg = (vertexId_t*) allocDeviceArray(batchsize, sizeof(vertexId_t));
	length_t* d_boff = (length_t*) allocDeviceArray(nv+1, sizeof(length_t));
	length_t* d_ell = (length_t*) allocDeviceArray(nv+1, sizeof(length_t));

	numBlocks.x = ceil((float)nv/(float)blockdim);
	initDeviceArray<<<numBlocks,blockdim>>>(d_ell, nv, 0);

	numBlocks.x = ceil((float)batchsize/(float)blockdim);
	calcEdgelistLengths<<<numBlocks,blockdim>>>(bu.getDeviceBUD()->devicePtr(), d_ell);

	thrust::device_ptr<vertexId_t> dp_ell(d_ell);
	thrust::device_ptr<vertexId_t> dp_boff(d_boff);
	thrust::exclusive_scan(dp_ell, dp_ell+nv+1, dp_boff);

	copyIndices<<<numBlocks,blockdim>>>(bu.getDeviceBUD()->devicePtr(), d_bind, d_bseg, d_boff, d_ell);

	numBlocks.x = ceil((float)nv/(float)blockdim);
	iSortAll<<<numBlocks,blockdim>>>(d_bind, d_boff, nv);
	printf("\n%s <%d> %f\n", __FUNCTION__, __LINE__, end_clock(ce_start, ce_stop));


	// MGPU segsort approach ========================================
	start_clock(ce_start, ce_stop);


	// mgpu::segmented_sort(d_bind, batchsize, d_boff+1, nv-2, mgpu::less_t<int>(), context);

	printf("\n%s <%d> %f\n", __FUNCTION__, __LINE__, end_clock(ce_start, ce_stop));


	// Thrust approach =============================================
	start_clock(ce_start, ce_stop);

	thrust::device_ptr<vertexId_t> dp_bind(bu.getDeviceBUD()->getDst());
	thrust::device_ptr<vertexId_t> dp_bseg(bu.getDeviceBUD()->getSrc());
	thrust::stable_sort_by_key(dp_bind, dp_bind + batchsize, dp_bseg);
	thrust::stable_sort_by_key(dp_bseg, dp_bseg + batchsize, dp_bind);	

	length_t* d_tell = (length_t*) allocDeviceArray(nv+1, sizeof(length_t));
	length_t* d_tboff = (length_t*) allocDeviceArray(nv+1, sizeof(length_t));

	numBlocks.x = ceil((float)nv/(float)blockdim);
	initDeviceArray<<<numBlocks,blockdim>>>(d_tell, nv, 0);

	numBlocks.x = ceil((float)batchsize/(float)blockdim);
	calcEdgelistLengths<<<numBlocks,blockdim>>>(bu.getDeviceBUD()->devicePtr(), d_tell);

	thrust::device_ptr<vertexId_t> dp_tell(d_tell);
	thrust::device_ptr<vertexId_t> dp_tboff(d_tboff);
	thrust::exclusive_scan(dp_tell, dp_tell+nv+1, dp_tboff);
	printf("\n%s <%d> %f\n", __FUNCTION__, __LINE__, end_clock(ce_start, ce_stop));


	// Correctness ==============================================

	// From iSort 
	vertexId_t* h_bind = (vertexId_t*) allocHostArray(batchsize, sizeof(vertexId_t));
	vertexId_t* h_bseg = (vertexId_t*) allocHostArray(batchsize, sizeof(vertexId_t));
	length_t* h_boff = (length_t*) allocHostArray(nv+1, sizeof(length_t));

	copyArrayDeviceToHost(d_bind, h_bind, batchsize, sizeof(vertexId_t));
	copyArrayDeviceToHost(d_bseg, h_bseg, batchsize, sizeof(vertexId_t));
	copyArrayDeviceToHost(d_boff, h_boff, nv, sizeof(length_t));

	// From Thrust
	vertexId_t* h_tbind = (vertexId_t*) allocHostArray(batchsize, sizeof(vertexId_t));
	vertexId_t* h_tbseg = (vertexId_t*) allocHostArray(batchsize, sizeof(vertexId_t));
	length_t* h_tboff = (length_t*) allocHostArray(nv+1, sizeof(length_t));

	copyArrayDeviceToHost(bu.getDeviceBUD()->getDst(), h_tbind, batchsize, sizeof(vertexId_t));
	copyArrayDeviceToHost(bu.getDeviceBUD()->getSrc(), h_tbseg, batchsize, sizeof(vertexId_t));
	copyArrayDeviceToHost(d_tboff, h_tboff, nv, sizeof(length_t));

	// Compare
	for (int i = 0; i < nv; ++i)
	{
		if (h_tboff[i] != h_boff[i])
		{
			printf("h_tboff = %d\t h_boff = %d\n", h_tboff[i], h_boff[i]);
		}
	}

	for (int i = 0; i < batchsize; ++i)
	{
		if (h_tbseg[i] != h_bseg[i])
		{
			printf("h_tbseg = %d\t h_bseg = %d\n", h_tbseg[i], h_bseg[i]);
		}
		if (h_tbind[i] != h_bind[i])
		{
			printf("h_tbind = %d\t h_bind = %d\n", h_tbind[i], h_bind[i]);
		}
	}
}

void testmgpusort(){
	mgpu::standard_context_t context;

	int count = 1000;
      int num_segments = div_up(count, 100);
      mem_t<int> segs = fill_random(0, count - 1, num_segments, true, context);
      std::vector<int> segs_host = from_mem(segs);
      mem_t<int> data = fill_random(0, 100000, count, false, context);
      mem_t<int> values(count, context);
      std::vector<int> host_data = from_mem(data);
      segmented_sort(data.data(), count, segs.data(), num_segments,
        less_t<int>(), context);
}

// TODO: change this into a CUDA mem copy operation.
__global__ void copyCSRToBUD(BatchUpdateData *bud, vertexId_t* const __restrict__ ind,
	vertexId_t* const __restrict__ seg)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	length_t batchSize = *(bud->getBatchSize());
	if (tid < batchSize)
	{
		bud->getSrc()[tid] = seg[tid];
		bud->getDst()[tid] = ind[tid];
	}
}

__global__ void copyOffCSRToBUD(BatchUpdateData *bud, length_t* const __restrict__ off)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	length_t nv = *(bud->getNumVertices());
	if (tid < nv+1)
	{
		bud->getOffsets()[tid] = off[tid];
	}
}

void sortBUD(length_t nv, BatchUpdate& bu,	const int blockdim)
{
	length_t batchsize = *(bu.getHostBUD()->getBatchSize());
	printf("batchsize %d\n", batchsize);

	dim3 numBlocks(1, 1);

	vertexId_t* d_bind = (vertexId_t*) allocDeviceArray(batchsize, sizeof(vertexId_t));
	length_t* d_boff = (length_t*) allocDeviceArray(nv+1, sizeof(length_t));
	vertexId_t* d_bseg = (vertexId_t*) allocDeviceArray(batchsize, sizeof(vertexId_t));
	length_t* d_ell = (length_t*) allocDeviceArray(nv+1, sizeof(length_t));

	numBlocks.x = ceil((float)nv/(float)blockdim);
	// TODO: use memset instead of this hack
	initDeviceArray<<<numBlocks,blockdim>>>(d_ell, nv, 0);
	initDeviceArray<<<numBlocks,blockdim>>>(d_boff, nv, 0);
	// TODO: find a home for this poor statement
	initDeviceArray<<<numBlocks,blockdim>>>(bu.getDeviceBUD()->getvNumDuplicates(), nv, 0);

	numBlocks.x = ceil((float)batchsize/(float)blockdim);
	calcEdgelistLengths<<<numBlocks,blockdim>>>(bu.getDeviceBUD()->devicePtr(), d_ell);

	thrust::device_ptr<vertexId_t> dp_ell(d_ell);
	thrust::device_ptr<vertexId_t> dp_boff(d_boff);
	thrust::exclusive_scan(dp_ell, dp_ell+nv+1, dp_boff);

	copyIndices<<<numBlocks,blockdim>>>(bu.getDeviceBUD()->devicePtr(), d_bind, d_bseg, d_boff, d_ell);

	numBlocks.x = ceil((float)nv/(float)blockdim);
	iSortAll<<<numBlocks,blockdim>>>(d_bind, d_boff, nv);

	// Put the sorted csr back into bud
	numBlocks.x = ceil((float)batchsize/(float)blockdim);
	copyCSRToBUD<<<numBlocks,blockdim>>>(bu.getDeviceBUD()->devicePtr(), d_bind, d_bseg);

	numBlocks.x = ceil((float)(nv+1)/(float)blockdim);
	copyOffCSRToBUD<<<numBlocks,blockdim>>>(bu.getDeviceBUD()->devicePtr(), d_boff);

	freeDeviceArray(d_bind);
	freeDeviceArray(d_boff);
	freeDeviceArray(d_bseg);
	freeDeviceArray(d_ell);
}

__global__ void comparecus(cuStinger* cus1, cuStinger* cus2)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	length_t nv = cus1->nv;
	if (tid < nv)
	{
		vertexId_t * adj1 = cus1->dVD->adj[tid]->dst;
		vertexId_t * adj2 = cus2->dVD->adj[tid]->dst;
		length_t size1 = cus1->dVD->getUsed()[tid];
		length_t size2 = cus2->dVD->getUsed()[tid];
		if (size1 != size2)
		{
			printf("size mismatch %d %d\n", size1, size2);
		}
		for (int i = 0; i < size1; ++i)
		{
			if (adj1[i] != adj2[i])
			{
				printf("adj mismatch vertex %d, %d %d\n", tid, adj1[i], adj2[i]);
				for (int j = 0; j < size1; ++j)
				{
					printf("%d adj1 %d\n", tid, adj1[j]);
				}
				printf("%d ==\n", tid);
				for (int j = 0; j < size1; ++j)
				{
					printf("%d adj2 %d\n", tid, adj2[j]);
				}
			}
		}
	}
}

void compareCUS(cuStinger* cus1, cuStinger* cus2)
{
	length_t nv = cus1->nv;

	dim3 numBlocks(1, 1);
	int32_t threads=32;
	dim3 threadsPerBlock(threads, 1);
	numBlocks.x = ceil((float)nv/(float)threads);
	comparecus<<<numBlocks, threadsPerBlock>>>(cus1->devicePtr(),cus2->devicePtr());
}
