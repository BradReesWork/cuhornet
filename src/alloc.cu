#include "hip/hip_runtime.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "main.h"

using namespace std;


void* allocHostArray(int32_t elements,int32_t eleSize){
	if (elements==0 || eleSize==0)
		return NULL;
	return malloc(eleSize*elements);
}

void* allocDeviceArray(int32_t elements,int32_t eleSize){
	int32_t* ptr=NULL;
	if (elements==0 || eleSize==0)
		return NULL;
	checkCudaErrors (hipMalloc((void **)&ptr,eleSize*elements));
	return ptr;
}

void freeHostArray(void* array){
	free(array);
}

void freeDeviceArray(void* array){
	checkCudaErrors(hipFree(array));
}

int32_t elementsPerVertex(int32_t elements){
	int32_t eleCount = elements;
	if(eleCount==0)
		eleCount=1;
	else if(eleCount < 5)
		eleCount*=2;
	else
		eleCount*=1.5;
	return eleCount;
}

void copyArrayHostToDevice(void* hostSrc, void* devDst, int32_t elements, int32_t eleSize){
	checkCudaErrors(hipMemcpy(devDst,hostSrc,elements*eleSize,hipMemcpyHostToDevice));
}

void copyArrayDeviceToHost(void* devSrc, void* hostDst, int32_t elements, int32_t eleSize){
	checkCudaErrors(hipMemcpy(hostDst,devSrc,elements*eleSize,hipMemcpyDeviceToHost));
	// cout << "D to H error : "<<  hipGetErrorString(code) << endl;
}

__global__ void devMakeGPUStinger(int32_t nv,int32_t ne,int32_t* d_off, int32_t* d_adj,
	int verticesPerThreadBlock,int32_t** d_adjArray, int32_t* d_adjSizeUsed){
	int32_t v_init=blockIdx.x*verticesPerThreadBlock;
	for (int v_hat=0; v_hat<verticesPerThreadBlock; v_hat++){
		int32_t v=v_init+v_hat;
		if(v>=nv)
			break;

		for(int32_t e=threadIdx.x; e<d_adjSizeUsed[v]; e+=blockDim.x){
			d_adjArray[v][e]=d_adj[d_off[v]+e];
		}

	}
}

void hostMakeGPUStinger(int32_t nv,int32_t ne,int32_t* h_off, int32_t* h_adj,
	int32_tPtr* d_adjArray,int32_t* d_adjSizeUsed,int32_t* d_adjSizeMax){

	int32_t* d_off = (int32_t*)allocDeviceArray(nv+1,sizeof(int32_t));
	int32_t* d_adj = (int32_t*)allocDeviceArray(ne,sizeof(int32_t));
	copyArrayHostToDevice(h_off,d_off,nv,sizeof(int32_t));
	copyArrayHostToDevice(h_adj,d_adj,nv,sizeof(int32_t));


	dim3 numBlocks(1, 1);
	int32_t threads=64;
	dim3 threadsPerBlock(threads, 1);

	numBlocks.x = ceil((float)nv/(float)threads);
	if (numBlocks.x>16000){
		numBlocks.x=16000;
	}	

	int32_t verticesPerThreadBlock = ceil(float(nv)/float(numBlocks.x-1));


	devMakeGPUStinger<<<numBlocks,threadsPerBlock>>>(nv,ne,d_off,d_adj,verticesPerThreadBlock,d_adjArray, d_adjSizeUsed);

	freeDeviceArray(d_adj);	
	freeDeviceArray(d_off);
}


void allocGPUMemory(int32_t nv,int32_t ne,int32_t* off, int32_t* adj,
	int32_tPtrPtr* d_adjArray,int32_t** d_adjSizeUsed,int32_t** d_adjSizeMax)
{	
	int32_tPtrPtr d_temp = (int32_t**)allocDeviceArray(nv,sizeof(int32_t*));
	*d_adjArray = d_temp;

	*d_adjSizeUsed = (int32_t*)allocDeviceArray(nv,sizeof(int32_t));
	*d_adjSizeMax =  (int32_t*)allocDeviceArray(nv,sizeof(int32_t));

	int32_tPtr* h_arrayPtr =  (int32_tPtr*)allocHostArray(nv,sizeof(int32_t*));
	int32_t* h_sizeArrayUsed =  (int32_t*)allocHostArray(nv,sizeof(int32_t));
	int32_t* h_sizeArrayMax =  (int32_t*)allocHostArray(nv,sizeof(int32_t));

	for(int v=0; v<nv; v++){
		h_sizeArrayUsed[v]=off[v+1]-off[v];
		h_sizeArrayMax[v] = elementsPerVertex(h_sizeArrayUsed[v]);
		h_arrayPtr[v] =  (int32_t*)allocDeviceArray(h_sizeArrayMax[v], sizeof(int32_t));
	}
	copyArrayHostToDevice(h_sizeArrayUsed,*d_adjSizeUsed,nv,sizeof(int32_t));
	copyArrayHostToDevice(h_sizeArrayMax,*d_adjSizeMax,nv,sizeof(int32_t));
	copyArrayHostToDevice(h_arrayPtr,*d_adjArray,nv,sizeof(int32_t*));

	freeHostArray(h_arrayPtr);
	freeHostArray(h_sizeArrayUsed);
	freeHostArray(h_sizeArrayMax);
}




