#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "main.hpp"

using namespace std;

__global__ void deviceEdgeDeletesSweep1(cuStinger* custing, BatchUpdateData* bud,int32_t updatesPerBlock){
	length_t* d_utilized      = custing->dVD->getUsed();
	length_t* d_max           = custing->dVD->getMax();
	cuStinger::cusEdgeData** d_adj = custing->dVD->getAdj();	
	vertexId_t* d_updatesSrc    = bud->getSrc();
	vertexId_t* d_updatesDst    = bud->getDst();
	length_t batchSize          = *(bud->getBatchSize());
	length_t* d_incCount        = bud->getIncCount();
	vertexId_t* d_indIncomplete = bud->getIndIncomplete();
	length_t* d_indDuplicate    = bud->getIndDuplicate();
	length_t* d_dupCount        = bud->getDuplicateCount();
	length_t* d_dupRelPos       = bud->getDupPosBatch();

	__shared__ int64_t found[1], research[1];

	int32_t init_pos = blockIdx.x * updatesPerBlock;

	// Updates are processed one at a time	
	for (int32_t i=0; i<updatesPerBlock; i++){
		int32_t pos=init_pos+i;
		if(pos>=batchSize)
			break;
		__syncthreads();


		vertexId_t src = d_updatesSrc[pos],dst = d_updatesDst[pos];

		if(threadIdx.x ==0){
			*found=-1;
		}

		__syncthreads();

		for(int iter=0; iter<10 && *found==-1; iter++){
			length_t srcInitSize = d_utilized[src];

			// Checking to see if the edge already exists in the graph. 
			for (length_t e=threadIdx.x; e<srcInitSize && *found==-1; e+=blockDim.x){
				if(d_adj[src]->dst[e]==dst){
					*found=e;
					break;
				}
			}
			__syncthreads();
	
			length_t ret;
			vertexId_t prevVal;

			if(*found!=-1 && threadIdx.x==0){
				prevVal = atomicCAS(d_adj[src]->dst + *found,dst,DELETION_MARKER);

				if(prevVal!=DELETION_MARKER){
					ret =  atomicSub(d_utilized+src, 1)-1;

					if(ret<0){
						int oldval = atomicAnd(d_utilized+src, 0);
					}
					else if(ret==0){
						d_adj[src]->dst[0]=DELETION_MARKER;						
					}
					else if(ret>0 && ret<=*found){
						d_adj[src]->dst[*found]=d_adj[src]->dst[ret];
						d_adj[src]->dst[ret]=DELETION_MARKER;
					}
				}
				else{
					*found=-1;
					printf("I beat to the deletion\n");
				}
			}
			// else if (*found==-1){
			// 	// Checking to see if the edge already exists in the graph. 
			// 	// for (length_t e=threadIdx.x; e<d_max[src]; e+=blockDim.x){
			// 	// 	if(d_adj[src]->dst[e]==dst){
			// 	// 		printf("AAGOOO\n");
			// 	// 		break;
			// 	// 	}
			// 	// }
			// }
			__syncthreads();
		}

	}
}


void cuStinger::edgeDeletions(BatchUpdate &bu)
{	
	dim3 numBlocks(1, 1);
	int32_t threads=32;
	dim3 threadsPerBlock(threads, 1);
	int32_t updatesPerBlock,dupsPerBlock;
	length_t updateSize,dupInBatch;

	updateSize = *(bu.getHostBUD()->getBatchSize());
	numBlocks.x = ceil((float)updateSize/(float)threads);
	if (numBlocks.x>16000){
		numBlocks.x=16000;
	}	
	updatesPerBlock = ceil(float(updateSize)/float(numBlocks.x));

	// cout << "Deletions : " << updatesPerBlock<< endl;
	// cout << "Deletions : " << numBlocks.x << endl;
	// cout << "Deletions : " << threadsPerBlock.x << endl;

	deviceEdgeDeletesSweep1<<<numBlocks,threadsPerBlock>>>(this->devicePtr(), bu.getDeviceBUD()->devicePtr(),updatesPerBlock);
	checkLastCudaError("Error in the first delete sweep");

	bu.getHostBUD()->copyDeviceToHost(*bu.getDeviceBUD());
	reAllocateMemoryAfterSweep1(bu);

	bu.getHostBUD()->resetIncCount();
	bu.getDeviceBUD()->resetIncCount();
	bu.getHostBUD()->resetDuplicateCount();
	bu.getDeviceBUD()->resetDuplicateCount();
}


	
__global__ void deviceVerifyDeletions(cuStinger* custing, BatchUpdateData* bud,int32_t updatesPerBlock, length_t* updateCounter){
	length_t* d_utilized      = custing->dVD->getUsed();
	length_t* d_max           = custing->dVD->getMax();
	cuStinger::cusEdgeData** d_adj = custing->dVD->getAdj();	
	vertexId_t* d_updatesSrc    = bud->getSrc();
	vertexId_t* d_updatesDst    = bud->getDst();
	length_t batchSize          = *(bud->getBatchSize());
	length_t* d_incCount        = bud->getIncCount();
	vertexId_t* d_indIncomplete = bud->getIndIncomplete();
	length_t* d_indDuplicate    = bud->getIndDuplicate();
	length_t* d_dupCount        = bud->getDuplicateCount();
	length_t* d_dupRelPos       = bud->getDupPosBatch();

	__shared__ int32_t found[1];

	int32_t init_pos = blockIdx.x * updatesPerBlock;

	if (threadIdx.x==0)
		updateCounter[blockIdx.x]=0;
	__syncthreads();

	// Updates are processed one at a time	
	for (int32_t i=0; i<updatesPerBlock; i++){
		int32_t pos=init_pos+i;
		if(pos>=batchSize)
			break;

		vertexId_t src = d_updatesSrc[pos],dst = d_updatesDst[pos];
		length_t srcInitSize = d_utilized[src];
		if(threadIdx.x ==0)
			*found=0;
		__syncthreads();

		// Checking to see if the edge already exists in the graph. 
		for (length_t e=threadIdx.x; e<srcInitSize && *found==0; e+=blockDim.x){
			if(d_adj[src]->dst[e]==dst || d_adj[src]->dst[e]==DELETION_MARKER){
				*found=1;
				break;
			}
			// if(d_adj[src]->dst[e]==DELETION_MARKER){
			// 	printf("DELETIONS are not smooth\n");
			// 	break;
			// }

		}
		__syncthreads();
	
		if (threadIdx.x==0)
			updateCounter[blockIdx.x]+=*found;
		__syncthreads();

	}
}


void cuStinger::verifyEdgeDeletions(BatchUpdate &bu)
{
	dim3 numBlocks(1, 1);
	int32_t threads=32;
	dim3 threadsPerBlock(threads, 1);
	int32_t updatesPerBlock,dupsPerBlock;
	length_t updateSize,dupInBatch;

	updateSize = *(bu.getHostBUD()->getBatchSize());
	numBlocks.x = ceil((float)updateSize/(float)threads);
	if (numBlocks.x>16000){
		numBlocks.x=16000;
	}	
	updatesPerBlock = ceil(float(updateSize)/float(numBlocks.x));

	length_t* devCounter = (length_t*)allocDeviceArray(numBlocks.x,sizeof(length_t));

	deviceVerifyDeletions<<<numBlocks,threadsPerBlock>>>(this->devicePtr(), bu.getDeviceBUD()->devicePtr(),updatesPerBlock,devCounter);

	length_t verified = cuStinger::sumDeviceArray(devCounter, numBlocks.x);

	if (verified==0)
		cout << "All deletions are accounted for.             Not deleted : " << verified << endl;
	else
		cout << "Some of the deletions are NOT accounted for. Not deleted : " << verified << endl;

	freeDeviceArray(devCounter);
}



