#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>


#include "main.h"

using namespace std;

__global__ void devUpdates(
	int32_tPtr* d_adjArray,int32_t* d_adjSizeUsed,int32_t* d_adjSizeMax,
	int32_t batchSize, int32_t updatesPerBlock ,int32_t* d_updatesSrc, int32_t* d_updatesDst, 
	int32_t* d_indIncomplete,int32_t* d_indCount)
{
	// if(threadIdx.x==0 && blockIdx.x==0)
	// 	printf("\nBatch size - \n%d\n", batchSize);
	int32_t init_pos = blockIdx.x * updatesPerBlock;
	// if(threadIdx.x==0 && blockIdx.x<100)
	// 	printf("%d %d %d\n",pos, blockIdx.x, updatesPerBlock);
	for(int i=threadIdx.x; i<updatesPerBlock; i+=blockDim.x){

		int32_t pos=init_pos+i;

		if(pos<batchSize){


			int32_t src = d_updatesSrc[pos];
			int32_t dst = d_updatesDst[pos];

			int32_t ret =  atomicAdd(d_adjSizeUsed+src, 1);

			if(ret<d_adjSizeMax[src]){
				d_adjArray[src][ret] = dst;
			}
			else{
				int32_t inCompleteEdgeID =  atomicAdd(d_indCount, 1);
				d_indIncomplete[inCompleteEdgeID] = pos;
				//RUN out of space
				// printf("%d %d \n", inCompleteEdgeID,pos);
				// printf("*");
			}
			d_updatesSrc[pos]=0;
			d_updatesDst[pos]=0;

		}
	}
}

// void update(int32_t nv,int32_t ne,
// 	int32_tPtr* d_adjArray,int32_t* d_adjSizeUsed,int32_t* d_adjSizeMax,
// 	int32_t numUpdates, int32_t* h_updatesSrc, int32_t* h_updatesDst, 
// 	int32_t* d_updatesSrc, int32_t* d_updatesDst)

void update(int32_t nv,int32_t ne,
	int32_tPtr* d_adjArray,int32_t* d_adjSizeUsed,int32_t* d_adjSizeMax,
	BatchUpdate &bu)
{	
	dim3 numBlocks(1, 1,1);
	int32_t threads=32;
	dim3 threadsPerBlock(threads, 1,1);

	int32_t batchSize = bu.getBatchSize();

	numBlocks.x = ceil((float)batchSize/(float)threads);
	if (numBlocks.x>65535){
		numBlocks.x=65535;
	}	
	int32_t updatesPerBatch = ceil(float(batchSize)/float(numBlocks.x-1));

	cout << numBlocks.x << " : " << threadsPerBlock.x << " : " << updatesPerBatch << endl;


	// devUpdates<<<100,threadsPerBlock>>>(d_adjArray,d_adjSizeUsed,d_adjSizeMax,
	// bu.getBatchSize(), bu.getDeviceSrcArray(), bu.getDeviceDstArray(),
	// bu.getDeviceIndInCompleteArray(), bu.getDeviceIndCount());

	devUpdates<<<numBlocks,threadsPerBlock>>>(d_adjArray,d_adjSizeUsed,d_adjSizeMax,
	bu.getBatchSize(), updatesPerBatch, bu.getDeviceSrcArray(), bu.getDeviceDstArray(),
	bu.getDeviceIndInCompleteArray(), bu.getDeviceIndCount());
	hipError_t error = hipGetLastError();
	if(error!=hipSuccess) {
		fprintf(stderr,"ERROR1: %s: %s\n", "Update error", hipGetErrorString(error) );
	}



}


BatchUpdate::BatchUpdate(int32_t batchSize_){
	batchSize=batchSize_;

	h_edgeSrc       =  (int32_t*)allocHostArray(batchSize,sizeof(int32_t));
	h_edgeDst       =  (int32_t*)allocHostArray(batchSize,sizeof(int32_t));
	h_indIncomplete =  (int32_t*)allocHostArray(batchSize,sizeof(int32_t));
	h_indCount      =  (int32_t*)allocHostArray(1,sizeof(int32_t));

	d_edgeSrc       =  (int32_t*)allocDeviceArray(batchSize,sizeof(int32_t));
	d_edgeDst       =  (int32_t*)allocDeviceArray(batchSize,sizeof(int32_t));
	d_indIncomplete =  (int32_t*)allocDeviceArray(batchSize,sizeof(int32_t));
	d_indCount      =  (int32_t*)allocDeviceArray(1,sizeof(int32_t));
}

BatchUpdate::~BatchUpdate(){
	freeHostArray(h_edgeSrc);
	freeHostArray(h_edgeDst);
	freeDeviceArray(d_edgeSrc);
	freeDeviceArray(d_edgeDst);
}

void BatchUpdate::copyHostToDevice(){
	copyArrayHostToDevice(h_edgeSrc, d_edgeSrc, batchSize, sizeof(int32_t));
	copyArrayHostToDevice(h_edgeDst, d_edgeDst, batchSize, sizeof(int32_t));
	copyArrayHostToDevice(h_indIncomplete, d_indIncomplete, batchSize, sizeof(int32_t));
	copyArrayHostToDevice(h_indCount, d_indCount, 1, sizeof(int32_t));
}

void BatchUpdate::copyDeviceToHost(){
	copyArrayDeviceToHost(d_edgeSrc, h_edgeSrc, batchSize, sizeof(int32_t));
	copyArrayDeviceToHost(d_edgeDst, h_edgeDst, batchSize, sizeof(int32_t));
	copyArrayDeviceToHost(d_indIncomplete, h_indIncomplete, batchSize, sizeof(int32_t));
	copyArrayDeviceToHost(d_indCount, h_indCount, 1, sizeof(int32_t));

}

