
#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    hipDeviceProp_t devive_prop;
    hipGetDeviceProperties(&devive_prop, 0);
    std::cout << devive_prop.major * 10 + devive_prop.minor
              << ";" << devive_prop.multiProcessorCount
              << ";" << devive_prop.name;
}
